// https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-generic-api-reference

hipsparseStatus_t
hipsparseCreateCsr(hipsparseSpMatDescr_t* spMatDescr,
                  int64_t               rows,
                  int64_t               cols,
                  int64_t               nnz,
                  void*                 csrRowOffsets,
                  void*                 csrColInd,
                  void*                 csrValues,
                  hipsparseIndexType_t   csrRowOffsetsType,
                  hipsparseIndexType_t   csrColIndType,
                  hipsparseIndexBase_t   idxBase,
                  hipDataType          valueType)

hipsparseStatus_t
hipsparseCreateConstCsr(hipsparseSpMatDescr_t* spMatDescr, //const descriptor
                       int64_t               rows,
                       int64_t               cols,
                       int64_t               nnz,
                       void*                 csrRowOffsets,
                       void*                 csrColInd,
                       void*                 csrValues,
                       hipsparseIndexType_t   csrRowOffsetsType,
                       hipsparseIndexType_t   csrColIndType,
                       hipsparseIndexBase_t   idxBase,
                       hipDataType          valueType)



hipsparseStatus_t
hipsparseSpMV_bufferSize(hipsparseHandle_t     handle,
                        hipsparseOperation_t  opA,
                        const void*          alpha,
                        hipsparseSpMatDescr_t matA, //const descriptor
                        hipsparseDnVecDescr_t vecX, //const descriptor
                        const void*          beta,
                        hipsparseDnVecDescr_t vecY,
                        hipDataType         computeType,
                        hipsparseSpMVAlg_t    alg,
                        size_t*              bufferSize)


hipsparseStatus_t
hipsparseSpMV(hipsparseHandle_t     handle,
             hipsparseOperation_t  opA,
             const void*          alpha,
             hipsparseSpMatDescr_t matA, //const descriptor
             hipsparseDnVecDescr_t vecX, //const descriptor
             const void*          beta,
             hipsparseDnVecDescr_t vecY,
             hipDataType         computeType,
             hipsparseSpMVAlg_t    alg,
             void*                externalBuffer)


https://docs.nvidia.com/cuda/cusparse/index.html#cusparsespgemm