
#include "sfem_cuda_blas.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <cassert>
#include <cstdio>

#define CHECK_CUDA(func)                                               \
    do {                                                               \
        hipError_t status = (func);                                   \
        if (status != hipSuccess) {                                   \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__,                                           \
                   hipGetErrorString(status),                         \
                   status);                                            \
            return EXIT_FAILURE;                                       \
        }                                                              \
    } while (0)

static const char *myCublasGetErrorString(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

#define CHECK_CUBLAS(func)                                               \
    do {                                                                 \
        hipblasStatus_t status = (func);                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                           \
            printf("CUBLAS API failed at line %d with error: %s (%d)\n", \
                   __LINE__,                                             \
                   myCublasGetErrorString(status),                       \
                   status);                                              \
            assert(false);                                               \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)

static bool cublas_initialized = false;
static hipblasHandle_t cublas_handle;
void __attribute__((destructor)) destroy_cublas() {
    if (cublas_initialized) {
        printf("Destroy CuBLAS\n");
        hipblasDestroy(cublas_handle);
    }
}

namespace sfem {
    namespace device {

        void cublas_init() {
            if (!cublas_initialized) {
                CHECK_CUBLAS(hipblasCreate(&cublas_handle));
            }
        }

        template <typename T>
        T *allocate(const std::size_t n) {
            T *ptr = nullptr;
            hipMalloc((void **)&ptr, n * sizeof(T));
            hipMemset(ptr, 0, n * sizeof(T));
            return ptr;
        }

        template <typename T>
        void destroy(T *a) {
            hipFree(a);
        }

        template <typename T>
        void copy(const ptrdiff_t n, const T *const src, T *const dest) {
            hipMemcpy(dest, src, n * sizeof(T), hipMemcpyDeviceToDevice);
        }

        double dot(const ptrdiff_t n, const double *const l, const double *const r) {
            cublas_init();

            double ret = 0;
            CHECK_CUBLAS(hipblasDdot(cublas_handle, n, l, 1, r, 1, &ret));
            return ret;
        }

        float dot(const ptrdiff_t n, const float *const l, const float *const r) {
            cublas_init();

            float ret = 0;
            CHECK_CUBLAS(hipblasSdot(cublas_handle, n, l, 1, r, 1, &ret));
            return ret;
        }

        void axpby(const ptrdiff_t n,
                   const double alpha,
                   const double *const x,
                   const double beta,
                   double *const y) {
            cublas_init();

            if (beta != 1) {
                CHECK_CUBLAS(hipblasDscal(cublas_handle, n, &beta, y, 1));
            }

            CHECK_CUBLAS(hipblasDaxpy(cublas_handle, n, &alpha, x, 1, y, 1));
        }

        void axpby(const ptrdiff_t n,
                   const float alpha,
                   const float *const x,
                   const float beta,
                   float *const y) {
            cublas_init();

            if (beta != 1) {
                CHECK_CUBLAS(hipblasSscal(cublas_handle, n, &beta, y, 1));
            }

            CHECK_CUBLAS(hipblasSaxpy(cublas_handle, n, &alpha, x, 1, y, 1));
        }

    }  // namespace device
}  // namespace sfem

extern "C" {

real_t *d_allocate(const std::size_t n) { return sfem::device::allocate<real_t>(n); }

void d_destroy(real_t *a) { sfem::device::destroy(a); }

void d_copy(const ptrdiff_t n, const real_t *const src, real_t *const dest) {
    sfem::device::copy(n, src, dest);
}

real_t d_dot(const ptrdiff_t n, const real_t *const l, const real_t *const r) {
    return sfem::device::dot(n, l, r);
}

void d_axpby(const ptrdiff_t n,
             const real_t alpha,
             const real_t *const x,
             const real_t beta,
             real_t *const y) {
    sfem::device::axpby(n, alpha, x, beta, y);
}

void d_memset(void *ptr, int value, const std::size_t n) { hipMemset(ptr, value, n); }
}
