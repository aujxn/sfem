#include "hip/hip_runtime.h"


#include <stdio.h>

#define real_t double
#define real_type real_t

#define MY_RESTRICT __restrict__

#define SFEM_USE_RULE56
#ifdef SFEM_USE_RULE56
#define TET4_NQP 56
static __device__ real_t tet4_qw[TET4_NQP] = {
        0.0010373112336140, 0.0010373112336140, 0.0010373112336140, 0.0010373112336140,
        0.0096016645399480, 0.0096016645399480, 0.0096016645399480, 0.0096016645399480,
        0.0096016645399480, 0.0096016645399480, 0.0096016645399480, 0.0096016645399480,
        0.0096016645399480, 0.0096016645399480, 0.0096016645399480, 0.0096016645399480,
        0.0164493976798232, 0.0164493976798232, 0.0164493976798232, 0.0164493976798232,
        0.0164493976798232, 0.0164493976798232, 0.0164493976798232, 0.0164493976798232,
        0.0164493976798232, 0.0164493976798232, 0.0164493976798232, 0.0164493976798232,
        0.0153747766513310, 0.0153747766513310, 0.0153747766513310, 0.0153747766513310,
        0.0153747766513310, 0.0153747766513310, 0.0153747766513310, 0.0153747766513310,
        0.0153747766513310, 0.0153747766513310, 0.0153747766513310, 0.0153747766513310,
        0.0293520118375230, 0.0293520118375230, 0.0293520118375230, 0.0293520118375230,
        0.0293520118375230, 0.0293520118375230, 0.0293520118375230, 0.0293520118375230,
        0.0293520118375230, 0.0293520118375230, 0.0293520118375230, 0.0293520118375230,
        0.0366291366405108, 0.0366291366405108, 0.0366291366405108, 0.0366291366405108};

static __device__ real_t tet4_qx[TET4_NQP] = {
        0.0149520651530592, 0.9551438045408220, 0.0149520651530592, 0.0149520651530592,
        0.1518319491659370, 0.7799760084415400, 0.0340960211962615, 0.0340960211962615,
        0.0340960211962615, 0.0340960211962615, 0.7799760084415400, 0.1518319491659370,
        0.7799760084415400, 0.1518319491659370, 0.0340960211962615, 0.0340960211962615,
        0.5526556431060171, 0.3549340560639790, 0.0462051504150017, 0.0462051504150017,
        0.0462051504150017, 0.0462051504150017, 0.3549340560639790, 0.5526556431060171,
        0.3549340560639790, 0.5526556431060171, 0.0462051504150017, 0.0462051504150017,
        0.2281904610687610, 0.5381043228880020, 0.2281904610687610, 0.2281904610687610,
        0.5381043228880020, 0.2281904610687610, 0.0055147549744775, 0.0055147549744775,
        0.0055147549744775, 0.5381043228880020, 0.2281904610687610, 0.2281904610687610,
        0.3523052600879940, 0.1961837595745600, 0.3523052600879940, 0.3523052600879940,
        0.1961837595745600, 0.3523052600879940, 0.0992057202494530, 0.0992057202494530,
        0.0992057202494530, 0.1961837595745600, 0.3523052600879940, 0.3523052600879940,
        0.1344783347929940, 0.5965649956210171, 0.1344783347929940, 0.1344783347929940};

static __device__ real_t tet4_qy[TET4_NQP] = {
        0.0149520651530592, 0.0149520651530592, 0.9551438045408220, 0.0149520651530592,
        0.0340960211962615, 0.0340960211962615, 0.1518319491659370, 0.7799760084415400,
        0.0340960211962615, 0.0340960211962615, 0.1518319491659370, 0.7799760084415400,
        0.0340960211962615, 0.0340960211962615, 0.7799760084415400, 0.1518319491659370,
        0.0462051504150017, 0.0462051504150017, 0.5526556431060171, 0.3549340560639790,
        0.0462051504150017, 0.0462051504150017, 0.5526556431060171, 0.3549340560639790,
        0.0462051504150017, 0.0462051504150017, 0.3549340560639790, 0.5526556431060171,
        0.2281904610687610, 0.2281904610687610, 0.5381043228880020, 0.0055147549744775,
        0.0055147549744775, 0.0055147549744775, 0.2281904610687610, 0.5381043228880020,
        0.2281904610687610, 0.2281904610687610, 0.5381043228880020, 0.2281904610687610,
        0.3523052600879940, 0.3523052600879940, 0.1961837595745600, 0.0992057202494530,
        0.0992057202494530, 0.0992057202494530, 0.3523052600879940, 0.1961837595745600,
        0.3523052600879940, 0.3523052600879940, 0.1961837595745600, 0.3523052600879940,
        0.1344783347929940, 0.1344783347929940, 0.5965649956210171, 0.1344783347929940};

static __device__ real_t tet4_qz[TET4_NQP] = {
        0.0149520651530592, 0.0149520651530592, 0.0149520651530592, 0.9551438045408220,
        0.0340960211962615, 0.0340960211962615, 0.0340960211962615, 0.0340960211962615,
        0.1518319491659370, 0.7799760084415400, 0.0340960211962615, 0.0340960211962615,
        0.1518319491659370, 0.7799760084415400, 0.1518319491659370, 0.7799760084415400,
        0.0462051504150017, 0.0462051504150017, 0.0462051504150017, 0.0462051504150017,
        0.5526556431060171, 0.3549340560639790, 0.0462051504150017, 0.0462051504150017,
        0.5526556431060171, 0.3549340560639790, 0.5526556431060171, 0.3549340560639790,
        0.0055147549744775, 0.0055147549744775, 0.0055147549744775, 0.2281904610687610,
        0.2281904610687610, 0.5381043228880020, 0.2281904610687610, 0.2281904610687610,
        0.5381043228880020, 0.2281904610687610, 0.2281904610687610, 0.5381043228880020,
        0.0992057202494530, 0.0992057202494530, 0.0992057202494530, 0.3523052600879940,
        0.3523052600879940, 0.1961837595745600, 0.3523052600879940, 0.3523052600879940,
        0.1961837595745600, 0.3523052600879940, 0.3523052600879940, 0.1961837595745600,
        0.1344783347929940, 0.1344783347929940, 0.1344783347929940, 0.5965649956210171};

#endif  // SFEM_USE_RULE56

////////////////////////////////////////////////////////
// tet4_transform_v2
////////////////////////////////////////////////////////
__device__ void tet4_transform_cu(
        /**
         ****************************************************************************************
        \begin{bmatrix}
        out_x \\
        out_y \\
        out_z
        \end{bmatrix}
        =
        \begin{bmatrix}
        px_0 \\
        py_0 \\
        pz_0
        \end{bmatrix}
        +
        \begin{bmatrix}
        px_1 - px_0 & px_2 - px_0 & px_3 - px_0 \\
        py_1 - py_0 & py_2 - py_0 & py_3 - py_0 \\
        pz_1 - pz_0 & pz_2 - pz_0 & pz_3 - pz_0
        \end{bmatrix}
        \cdot
        \begin{bmatrix}
        qx \\
        qy \\
        qz
        \end{bmatrix}
        *************************************************************************************************
      */

        // X-coordinates
        const real_type px0,
        const real_type px1,
        const real_type px2,
        const real_type px3,
        // Y-coordinates
        const real_type py0,
        const real_type py1,
        const real_type py2,
        const real_type py3,
        // Z-coordinates
        const real_type pz0,
        const real_type pz1,
        const real_type pz2,
        const real_type pz3,
        // Quadrature point
        const real_type qx,
        const real_type qy,
        const real_type qz,
        // Output
        real_type* const out_x,
        real_type* const out_y,
        real_type* const out_z) {
    //
    //
    *out_x = px0 + qx * (-px0 + px1) + qy * (-px0 + px2) + qz * (-px0 + px3);
    *out_y = py0 + qx * (-py0 + py1) + qy * (-py0 + py2) + qz * (-py0 + py3);
    *out_z = pz0 + qx * (-pz0 + pz1) + qy * (-pz0 + pz2) + qz * (-pz0 + pz3);
}

////////////////////////////////////////////////////////
// tet4_measure_v2
////////////////////////////////////////////////////////
__device__ real_type tet4_measure_cu(
        // X-coordinates
        const real_type px0,
        const real_type px1,
        const real_type px2,
        const real_type px3,
        // Y-coordinates
        const real_type py0,
        const real_type py1,
        const real_type py2,
        const real_type py3,
        // Z-coordinates
        const real_type pz0,
        const real_type pz1,
        const real_type pz2,
        const real_type pz3) {
    //
    // determinant of the Jacobian
    // M = [px0, py0, pz0, 1]
    //     [px1, py1, pz1, 1]
    //     [px2, py2, pz2, 1]
    //     [px3, py3, pz3, 1]
    //
    // V = (1/6) * det(M)

    const real_type x0 = -pz0 + pz3;
    const real_type x1 = -py0 + py2;
    const real_type x2 = -(1.0 / 6.0) * px0 + (1.0 / 6.0) * px1;
    const real_type x3 = -py0 + py3;
    const real_type x4 = -pz0 + pz2;
    const real_type x5 = -py0 + py1;
    const real_type x6 = -(1.0 / 6.0) * px0 + (1.0 / 6.0) * px2;
    const real_type x7 = -pz0 + pz1;
    const real_type x8 = -(1.0 / 6.0) * px0 + (1.0 / 6.0) * px3;

    return x0 * x1 * x2 - x0 * x5 * x6 - x1 * x7 * x8 - x2 * x3 * x4 + x3 * x6 * x7 + x4 * x5 * x8;
}

////////////////////////////////////////////////////////
// hex_aa_8_eval_fun_V
////////////////////////////////////////////////////////
__device__ void hex_aa_8_eval_fun_cu(
        // Quadrature point (local coordinates)
        // With respect to the hat functions of a cube element
        // In a local coordinate system
        const real_t x,
        const real_t y,
        const real_t z,

        // Output
        real_t* const MY_RESTRICT f0,
        real_t* const MY_RESTRICT f1,
        real_t* const MY_RESTRICT f2,
        real_t* const MY_RESTRICT f3,
        real_t* const MY_RESTRICT f4,
        real_t* const MY_RESTRICT f5,
        real_t* const MY_RESTRICT f6,
        real_t* const MY_RESTRICT f7) {
    //
    *f0 = (1.0 - x) * (1.0 - y) * (1.0 - z);
    *f1 = x * (1.0 - y) * (1.0 - z);
    *f2 = x * y * (1.0 - z);
    *f3 = (1.0 - x) * y * (1.0 - z);
    *f4 = (1.0 - x) * (1.0 - y) * z;
    *f5 = x * (1.0 - y) * z;
    *f6 = x * y * z;
    *f7 = (1.0 - x) * y * z;
}

////////////////////////////////////////////////////////
// hex_aa_8_collect_coeffs_cu
////////////////////////////////////////////////////////
__device__ void hex_aa_8_collect_coeffs_cu(
        //
        const ptrdiff_t* const MY_RESTRICT stride,
        const ptrdiff_t i,
        const ptrdiff_t j,
        const ptrdiff_t k,
        // Attention this is geometric data transformed to solver data!
        const real_t* const MY_RESTRICT data,
        //
        real_t* MY_RESTRICT out0,
        real_t* MY_RESTRICT out1,
        real_t* MY_RESTRICT out2,
        real_t* MY_RESTRICT out3,
        real_t* MY_RESTRICT out4,
        real_t* MY_RESTRICT out5,
        real_t* MY_RESTRICT out6,
        real_t* MY_RESTRICT out7) {
    //
    const ptrdiff_t i0 = i * stride[0] + j * stride[1] + k * stride[2];
    const ptrdiff_t i1 = (i + 1) * stride[0] + j * stride[1] + k * stride[2];
    const ptrdiff_t i2 = (i + 1) * stride[0] + (j + 1) * stride[1] + k * stride[2];
    const ptrdiff_t i3 = i * stride[0] + (j + 1) * stride[1] + k * stride[2];
    const ptrdiff_t i4 = i * stride[0] + j * stride[1] + (k + 1) * stride[2];
    const ptrdiff_t i5 = (i + 1) * stride[0] + j * stride[1] + (k + 1) * stride[2];
    const ptrdiff_t i6 = (i + 1) * stride[0] + (j + 1) * stride[1] + (k + 1) * stride[2];
    const ptrdiff_t i7 = i * stride[0] + (j + 1) * stride[1] + (k + 1) * stride[2];

    *out0 = data[i0];
    *out1 = data[i1];
    *out2 = data[i2];
    *out3 = data[i3];
    *out4 = data[i4];
    *out5 = data[i5];
    *out6 = data[i6];
    *out7 = data[i7];
}

////////////////////////////////////////////////////////
// hex_aa_8_collect_coeffs_cu
////////////////////////////////////////////////////////
__device__ void hex_aa_8_collect_coeffs_cu_v2(
        //
        const ptrdiff_t MY_RESTRICT stride0,
        const ptrdiff_t MY_RESTRICT stride1,
        const ptrdiff_t MY_RESTRICT stride2,

        const ptrdiff_t i,
        const ptrdiff_t j,
        const ptrdiff_t k,
        // Attention this is geometric data transformed to solver data!
        const real_t* const MY_RESTRICT data,
        //
        real_t* MY_RESTRICT out0,
        real_t* MY_RESTRICT out1,
        real_t* MY_RESTRICT out2,
        real_t* MY_RESTRICT out3,
        real_t* MY_RESTRICT out4,
        real_t* MY_RESTRICT out5,
        real_t* MY_RESTRICT out6,
        real_t* MY_RESTRICT out7) {
    //
    const ptrdiff_t i0 = i * stride0 + j * stride1 + k * stride2;
    const ptrdiff_t i1 = (i + 1) * stride0 + j * stride1 + k * stride2;
    const ptrdiff_t i2 = (i + 1) * stride0 + (j + 1) * stride1 + k * stride2;
    const ptrdiff_t i3 = i * stride0 + (j + 1) * stride1 + k * stride2;
    const ptrdiff_t i4 = i * stride0 + j * stride1 + (k + 1) * stride2;
    const ptrdiff_t i5 = (i + 1) * stride0 + j * stride1 + (k + 1) * stride2;
    const ptrdiff_t i6 = (i + 1) * stride0 + (j + 1) * stride1 + (k + 1) * stride2;
    const ptrdiff_t i7 = i * stride0 + (j + 1) * stride1 + (k + 1) * stride2;

    *out0 = data[i0];
    *out1 = data[i1];
    *out2 = data[i2];
    *out3 = data[i3];
    *out4 = data[i4];
    *out5 = data[i5];
    *out6 = data[i6];
    *out7 = data[i7];
}

// Struct for the elements
typedef struct {
    int* elems_v0;
    int* elems_v1;
    int* elems_v2;
    int* elems_v3;
} elems_tet4_device;

void                                                              //
cuda_allocate_elems_tet4_device(elems_tet4_device* elems_device,  //
                                const ptrdiff_t nelements) {      //
    hipMalloc((void**)&elems_device->elems_v0, nelements * sizeof(int));
    hipMalloc((void**)&elems_device->elems_v1, nelements * sizeof(int));
    hipMalloc((void**)&elems_device->elems_v2, nelements * sizeof(int));
    hipMalloc((void**)&elems_device->elems_v3, nelements * sizeof(int));
}

void free_elems_tet4_device(elems_tet4_device* elems_device) {
    hipFree(elems_device->elems_v0);
    hipFree(elems_device->elems_v1);
    hipFree(elems_device->elems_v2);
    hipFree(elems_device->elems_v3);
}

// Struct for xyz
typedef struct {
    float* x;
    float* y;
    float* z;
} xyz_tet4_device;

void cuda_allocate_xyz_tet4_device(xyz_tet4_device* xyz_device, const ptrdiff_t nnodes) {
    hipMalloc((void**)&xyz_device->x, nnodes * sizeof(float));
    hipMalloc((void**)&xyz_device->y, nnodes * sizeof(float));
    hipMalloc((void**)&xyz_device->z, nnodes * sizeof(float));
}

void free_xyz_tet4_device(xyz_tet4_device* xyz_device) {
    hipFree(xyz_device->x);
    hipFree(xyz_device->y);
    hipFree(xyz_device->z);
}

//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
// tet4_resample_field_local_kernel //////////////////////
//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
__global__ void tet4_resample_field_local_kernel(
        // Mesh
        const ptrdiff_t start_element,
        const ptrdiff_t end_element,
        const ptrdiff_t nnodes,
        const elems_tet4_device MY_RESTRICT elems,
        const xyz_tet4_device MY_RESTRICT xyz,
        // SDF
        // const ptrdiff_t* const MY_RESTRICT n,
        const ptrdiff_t MY_RESTRICT stride0,
        const ptrdiff_t MY_RESTRICT stride1,
        const ptrdiff_t MY_RESTRICT stride2,

        const float* const MY_RESTRICT origin,
        const float* const MY_RESTRICT delta,
        const real_type* const MY_RESTRICT data,
        // Output
        real_type* const MY_RESTRICT weighted_field) {
    //
    // Thread index
    const ptrdiff_t element_i = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("element_i = %ld\n", element_i);

    if (element_i < start_element || element_i >= end_element) {
        return;
    }

    // weighted_field[element_i] = 11.0;

    ////////////////////////////////////////////////////

    const real_type ox = (real_type)origin[0];
    const real_type oy = (real_type)origin[1];
    const real_type oz = (real_type)origin[2];

    const real_type dx = (real_type)delta[0];
    const real_type dy = (real_type)delta[1];
    const real_type dz = (real_type)delta[2];

    ////////////////////////////////////////////////////

    real_type x0 = 0.0, x1 = 0.0, x2 = 0.0, x3 = 0.0;
    real_type y0 = 0.0, y1 = 0.0, y2 = 0.0, y3 = 0.0;
    real_type z0 = 0.0, z1 = 0.0, z2 = 0.0, z3 = 0.0;

    // real_type hex8_f[8];
    real_type hex8_f0 = 0.0, hex8_f1 = 0.0, hex8_f2 = 0.0, hex8_f3 = 0.0, hex8_f4 = 0.0,
              hex8_f5 = 0.0, hex8_f6 = 0.0, hex8_f7 = 0.0;

    // real_type coeffs[8];
    real_type coeffs0 = 0.0, coeffs1 = 0.0, coeffs2 = 0.0, coeffs3 = 0.0, coeffs4 = 0.0,
              coeffs5 = 0.0, coeffs6 = 0.0, coeffs7 = 0.0;

    // real_type tet4_f[4];
    real_type tet4_f0 = 0.0, tet4_f1 = 0.0, tet4_f2 = 0.0, tet4_f3 = 0.0;

    // real_type element_field[4];
    real_type element_field0 = 0.0, element_field1 = 0.0, element_field2 = 0.0,
              element_field3 = 0.0;

    // loop over the 4 vertices of the tetrahedron
    int ev[4];
    ev[0] = elems.elems_v0[element_i];
    ev[1] = elems.elems_v1[element_i];
    ev[2] = elems.elems_v2[element_i];
    ev[3] = elems.elems_v3[element_i];

    // for (int v = 0; v < 4; ++v) {
    //     ev[v] = elems[v][element_i];
    // }

    // copy the coordinates of the vertices
    // for (int v = 0; v < 4; ++v) {
    //     x[v] = xyz[0][ev[v]];  // x-coordinates
    //     y[v] = xyz[1][ev[v]];  // y-coordinates
    //     z[v] = xyz[2][ev[v]];  // z-coordinates
    // }
    {
        // x0 = xyz[0][ev[0]];
        // x1 = xyz[0][ev[1]];
        // x2 = xyz[0][ev[2]];
        // x3 = xyz[0][ev[3]];

        // y0 = xyz[1][ev[0]];
        // y1 = xyz[1][ev[1]];
        // y2 = xyz[1][ev[2]];
        // y3 = xyz[1][ev[3]];

        // z0 = xyz[2][ev[0]];
        // z1 = xyz[2][ev[1]];
        // z2 = xyz[2][ev[2]];
        // z3 = xyz[2][ev[3]];

        x0 = xyz.x[ev[0]];
        x1 = xyz.x[ev[1]];
        x2 = xyz.x[ev[2]];
        x3 = xyz.x[ev[3]];

        y0 = xyz.y[ev[0]];
        y1 = xyz.y[ev[1]];
        y2 = xyz.y[ev[2]];
        y3 = xyz.y[ev[3]];

        z0 = xyz.z[ev[0]];
        z1 = xyz.z[ev[1]];
        z2 = xyz.z[ev[2]];
        z3 = xyz.z[ev[3]];
    }

    // Volume of the tetrahedron
    const real_type theta_volume = tet4_measure_cu(x0,
                                                   x1,
                                                   x2,
                                                   x3,
                                                   //
                                                   y0,
                                                   y1,
                                                   y2,
                                                   y3,
                                                   //
                                                   z0,
                                                   z1,
                                                   z2,
                                                   z3);

    /////////////////////////////////////////////
    // loop over the quadrature points
    for (int quad_i = 0; quad_i < TET4_NQP; quad_i++) {  // loop over the quadrature points

        real_type g_qx, g_qy, g_qz;

        tet4_transform_cu(x0,
                          x1,
                          x2,
                          x3,

                          y0,
                          y1,
                          y2,
                          y3,

                          z0,
                          z1,
                          z2,
                          z3,

                          tet4_qx[quad_i],
                          tet4_qy[quad_i],
                          tet4_qz[quad_i],

                          &g_qx,
                          &g_qy,
                          &g_qz);

#ifdef SFEM_RESAMPLE_GAP_DUAL
        // Standard basis function
        {
            tet4_f[0] = 1 - tet4_qx[q] - tet4_qy[q] - tet4_qz[q];
            tet4_f[1] = tet4_qx[q];
            tet4_f[2] = tet4_qy[q];
            tet4_f[2] = tet4_qz[q];
        }
#else
        // DUAL basis function
        {
            const real_type f0 = 1.0 - tet4_qx[quad_i] - tet4_qy[quad_i] - tet4_qz[quad_i];
            const real_type f1 = tet4_qx[quad_i];
            const real_type f2 = tet4_qy[quad_i];
            const real_type f3 = tet4_qz[quad_i];

            tet4_f0 = 4.0 * f0 - f1 - f2 - f3;
            tet4_f1 = -f0 + 4.0 * f1 - f2 - f3;
            tet4_f2 = -f0 - f1 + 4.0 * f2 - f3;
            tet4_f3 = -f0 - f1 - f2 + 4.0 * f3;
        }
#endif

        const real_type grid_x = (g_qx - ox) / dx;
        const real_type grid_y = (g_qy - oy) / dy;
        const real_type grid_z = (g_qz - oz) / dz;

        const ptrdiff_t i = floor(grid_x);
        const ptrdiff_t j = floor(grid_y);
        const ptrdiff_t k = floor(grid_z);

        // Get the reminder [0, 1]
        real_type l_x = (grid_x - (double)i);
        real_type l_y = (grid_y - (double)j);
        real_type l_z = (grid_z - (double)k);

        // Critical point
        hex_aa_8_eval_fun_cu(l_x,
                             l_y,
                             l_z,
                             &hex8_f0,
                             &hex8_f1,
                             &hex8_f2,
                             &hex8_f3,
                             &hex8_f4,
                             &hex8_f5,
                             &hex8_f6,
                             &hex8_f7);

        hex_aa_8_collect_coeffs_cu_v2(stride0,
                                      stride1,
                                      stride2,
                                      i,
                                      j,
                                      k,
                                      data,
                                      &coeffs0,
                                      &coeffs1,
                                      &coeffs2,
                                      &coeffs3,
                                      &coeffs4,
                                      &coeffs5,
                                      &coeffs6,
                                      &coeffs7);

        // Integrate gap function
        {
            real_type eval_field = 0.0;
            // UNROLL_ZERO
            // for (int edof_j = 0; edof_j < 8; edof_j++) {
            //     eval_field += hex8_f[edof_j] * coeffs[edof_j];
            // }
            eval_field += hex8_f0 * coeffs0;
            eval_field += hex8_f1 * coeffs1;
            eval_field += hex8_f2 * coeffs2;
            eval_field += hex8_f3 * coeffs3;
            eval_field += hex8_f4 * coeffs4;
            eval_field += hex8_f5 * coeffs5;
            eval_field += hex8_f6 * coeffs6;
            eval_field += hex8_f7 * coeffs7;

            // UNROLL_ZERO
            // for (int edof_i = 0; edof_i < 4; edof_i++) {
            //     element_field[edof_i] += eval_field * tet4_f[edof_i] * dV;
            // }  // end edof_i loop

            real_type dV = theta_volume * tet4_qw[quad_i];
            // dV = 1.0;

            element_field0 += eval_field * tet4_f0 * dV;
            element_field1 += eval_field * tet4_f1 * dV;
            element_field2 += eval_field * tet4_f2 * dV;
            element_field3 += eval_field * tet4_f3 * dV;

        }  // end integrate gap function

    }  // end loop over the quadrature points

    atomicAdd(&weighted_field[ev[0]], element_field0);
    atomicAdd(&weighted_field[ev[1]], element_field1);
    atomicAdd(&weighted_field[ev[2]], element_field2);
    atomicAdd(&weighted_field[ev[3]], element_field3);

}  // end kernel tet4_resample_field_local_CU

__global__ void mykernel() { printf("hello fron kernel\n"); }

//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
// tet4_resample_field_local_v2 //////////////////////////
//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
/*
nvcc --gpu-architecture=sm_75 --device-c sfem_resample_field_cuda.cu -o sfem_resample_field_cuda.o
nvcc -O3 --gpu-architecture=sm_75 --shared --compiler-options -fPIC -shared
sfem_resample_field_cuda.cu -o sfem_resample_field_cuda.so


nvcc -O3  --gpu-architecture=sm_75 --compiler-options -fPIC -shared sfem_resample_field_cuda.cu -o
sfem_resample_field_cuda.o ar -r sfem_resample_field_cuda.a sfem_resample_field_cuda.o

*/

extern "C" int tet4_resample_field_local_CUDA(
        // Mesh
        const ptrdiff_t nelements,
        const ptrdiff_t nnodes,
        int** const MY_RESTRICT elems,
        float** const MY_RESTRICT xyz,
        // SDF
        const ptrdiff_t* const MY_RESTRICT n,
        const ptrdiff_t* const MY_RESTRICT stride,
        const float* const MY_RESTRICT origin,
        const float* const MY_RESTRICT delta,
        const real_type* const MY_RESTRICT data,
        // Output
        real_type* const MY_RESTRICT weighted_field) {
    //

    printf("=============================================\n");
    printf("nelements = %ld\n", nelements);
    printf("=============================================\n");

    // Allocate memory on the device

    // Allocate weighted_field on the device
    double* weighted_field_device;
    hipMalloc((void**)&weighted_field_device, nnodes * sizeof(double));
    hipMemset(weighted_field_device, 0, sizeof(double) * nnodes);

    // copy the elements to the device
    elems_tet4_device elems_device;
    cuda_allocate_elems_tet4_device(&elems_device, nelements);

    hipMemcpy(elems_device.elems_v0, elems[0], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(elems_device.elems_v1, elems[1], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(elems_device.elems_v2, elems[2], nelements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(elems_device.elems_v3, elems[3], nelements * sizeof(int), hipMemcpyHostToDevice);

    // Allocate xyz on the device
    xyz_tet4_device xyz_device;
    hipMalloc((void**)&xyz_device, 3 * sizeof(float*));
    cuda_allocate_xyz_tet4_device(&xyz_device, nnodes);
    hipMemcpy(xyz_device.x, xyz[0], nnodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(xyz_device.y, xyz[1], nnodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(xyz_device.z, xyz[2], nnodes * sizeof(float), hipMemcpyHostToDevice);

    // ptrdiff_t* stride_device;
    // hipMalloc((void**)&stride_device, 3 * sizeof(ptrdiff_t));
    // hipMemcpy(stride_device, stride, 3 * sizeof(ptrdiff_t), hipMemcpyHostToDevice);

    float* origin_device;
    hipMalloc((void**)&origin_device, 3 * sizeof(float));
    hipMemcpy(origin_device, origin, 3 * sizeof(float), hipMemcpyHostToDevice);

    float* delta_device;
    hipMalloc((void**)&delta_device, 3 * sizeof(float));
    hipMemcpy(delta_device, delta, 3 * sizeof(float), hipMemcpyHostToDevice);

    double* data_device;
    const ptrdiff_t size_data = n[0] * n[1] * n[2];
    hipMalloc((void**)&data_device, size_data * sizeof(double));
    hipMemcpy(data_device, data, size_data * sizeof(double), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    ///////////////////////////////////////////////////////////////////////////////
    // Call the kernel
    hipEvent_t start, stop;

    // Number of threads
    const ptrdiff_t threadsPerBlock = 128;

    // Number of blocks
    const ptrdiff_t numBlocks = (nelements + threadsPerBlock - 1) / threadsPerBlock;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("============================================================================\n");
    printf("GPU:    Launching the kernel\n");
    printf("GPU:    Number of blocks:            %ld\n", numBlocks);
    printf("GPU:    Number of threads per block: %ld\n", threadsPerBlock);
    printf("GPU:    Total number of threads:     %ld\n", (numBlocks * threadsPerBlock));
    printf("============================================================================\n");

    hipEventRecord(start);

    tet4_resample_field_local_kernel<<<numBlocks, threadsPerBlock>>>(0,             //
                                                                     nelements,     //
                                                                     nnodes,        //
                                                                     elems_device,  //
                                                                     xyz_device,
                                                                     //  NULL,
                                                                     stride[0],
                                                                     stride[1],
                                                                     stride[2],
                                                                     origin_device,
                                                                     delta_device,
                                                                     data_device,
                                                                     weighted_field_device);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // get cuda error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(error));
    }

    // end kernel
    ///////////////////////////////////////////////////////////////////////////////

    double time = milliseconds / 1000.0;

    const double elements_second = (double)nelements / time;

    printf("============================================================================\n");
    printf("GPU:    Elapsed time: %e s\n", time);
    printf("GPU:    Throughput:   %e elements/second\n", elements_second);
    printf("============================================================================\n");

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free memory on the device
    free_elems_tet4_device(&elems_device);

    free_xyz_tet4_device(&xyz_device);

    // hipFree(stride_device);
    hipFree(origin_device);
    hipFree(delta_device);

    // Copy the result back to the host
    hipMemcpy(weighted_field,           //
               weighted_field_device,    //
               nnodes * sizeof(double),  //
               hipMemcpyDeviceToHost);  //

    hipFree(weighted_field_device);

    hipFree(data_device);

    return 0;
}