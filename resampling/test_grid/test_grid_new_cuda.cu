#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2:  // Fermi
            if (devProp.minor == 1)
                cores = mp * 48;
            else
                cores = mp * 32;
            break;
        case 3:  // Kepler
            cores = mp * 192;
            break;
        case 5:  // Maxwell
            cores = mp * 128;
            break;
        case 6:  // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2))
                cores = mp * 128;
            else if (devProp.minor == 0)
                cores = mp * 64;
            else
                printf("Unknown device type\n");
            break;
        case 7:  // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5))
                cores = mp * 64;
            else
                printf("Unknown device type\n");
            break;
        case 8:  // Ampere
            if (devProp.minor == 0)
                cores = mp * 64;
            else if (devProp.minor == 6)
                cores = mp * 128;
            else if (devProp.minor == 9)
                cores = mp * 128;  // ada lovelace
            else
                printf("Unknown device type\n");
            break;
        case 9:  // Hopper
            if (devProp.minor == 0)
                cores = mp * 128;
            else
                printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

#include "test_grid_new.h"

/**
 * @brief Calculates the nearest grid coordinates (floor values) for a given point.
 *
 * This function calculates the nearest grid coordinates (floor values) for a given point (x, y)
 * based on the grid origin (x_zero, y_zero) and grid spacing (delta_x, delta_y). The calculated
 * grid coordinates are stored in the variables i and j.
 *
 * @param x_zero The x-coordinate of the grid origin.
 * @param y_zero The y-coordinate of the grid origin.
 * @param delta_x The spacing between grid points along the x-axis.
 * @param delta_y The spacing between grid points along the y-axis.
 * @param x The x-coordinate of the point for which nearest grid coordinates are to be calculated.
 * @param y The y-coordinate of the point for which nearest grid coordinates are to be calculated.
 * @param i Reference to the variable where the calculated x-coordinate of the nearest grid point
 * will be stored.
 * @param j Reference to the variable where the calculated y-coordinate of the nearest grid point
 * will be stored.
 */
__device__ void get_nearest_coordinates_floor_cu(const double x_zero,
                                                 const double y_zero,
                                                 const double delta_x,
                                                 const double delta_y,
                                                 const double x,
                                                 const double y,
                                                 int& i,
                                                 int& j) {
    //
    i = static_cast<int>(floor((x - x_zero) / delta_x));
    j = static_cast<int>(floor((y - y_zero) / delta_y));
}

/**
 * @brief Calculates the domain boundaries for a given domain number within a stripe.
 *
 * This function calculates the minimum and maximum values of the x and y coordinates
 * for a specific domain within a stripe. The domain number is used to determine the
 * position of the domain within the stripe.
 *
 * @param ds The domains_stripe struct containing information about the stripe.
 * @param domain_nr The number of the domain within the stripe.
 * @param x_min The minimum x coordinate of the domain.
 * @param y_min The minimum y coordinate of the domain.
 * @param x_max The maximum x coordinate of the domain.
 * @param y_max The maximum y coordinate of the domain.
 */
__device__ void get_domain_from_stripe_cu(const domains_stripe& ds,
                                          const size_t domain_nr,
                                          double& x_min,
                                          double& y_min,
                                          double& x_max,
                                          double& y_max) {
    //
    x_min = ds.x_min + domain_nr * ds.side_x;
    y_min = ds.y_min;

    x_max = x_min + ds.side_x;
    y_max = y_min + ds.side_y;
}

/**
 * @brief Performs the quadrature for a single stripe.
 *
 * @param Qs Pointer to the array where the calculated quadrature values will be stored.
 * @param gg The global_grid_type struct containing information about the global grid.
 * @param qr The quadrature_rule struct containing information about the quadrature rule.
 * @param qr_nodes_nr_  The number of nodes in the quadrature rule.
 * @param ds The domains_stripe struct containing information about the stripe.
 * @return void
 */
__device__ void perform_quadrature_global_stripe(double* Qs,                  //
                                                 const global_grid_type& gg,  //
                                                 const quadrature_rule& qr,   //
                                                 const size_t qr_nodes_nr_,   //
                                                 const domains_stripe& ds) {  //
    //
    // const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t domain_nr = blockIdx.x * blockDim.x + threadIdx.x;
    // const size_t stripe_nr = blockIdx.x;

    if (domain_nr >= ds.nr_domains) {
        return;
    }

    // double Ql = 0.0;

    // for (size_t i = 0; i < ds.nr_domains; ++i) {
    double x_d_min, y_d_min, x_d_max, y_d_max;
    get_domain_from_stripe_cu(ds, domain_nr, x_d_min, y_d_min, x_d_max, y_d_max);

    const double volume = (x_d_max - x_d_min) * (y_d_max - y_d_min);  // 3 * ds_nr_domains

    double Qs_i = 0.0;

    const size_t qr_size = qr_nodes_nr_;

    for (size_t q_i = 0; q_i < qr_size; ++q_i) {
        //
        // 2 * 3 * qr_size * ds.nr_domains
        const double x_Q = (qr.x_nodes_ptr_cu[q_i]) * (x_d_max - x_d_min) + x_d_min;
        const double y_Q = (qr.y_nodes_ptr_cu[q_i]) * (y_d_max - y_d_min) + y_d_min;

        // std::cout << "x_Q: " << x_Q << " y_Q: " << y_Q << std::endl;

        int i_local, j_local;

        get_nearest_coordinates_floor_cu(gg.x_zero,  //
                                         gg.y_zero,  //
                                         gg.delta,
                                         gg.delta,
                                         x_Q,
                                         y_Q,
                                         i_local,
                                         j_local);

        // data trasfer 4 * 8 * qr_size * dsnr_domains
        const double f1 = gg.grid_ptr_cu[i_local * gg.x_size + j_local];
        const double f2 = gg.grid_ptr_cu[i_local * gg.y_size + j_local + 1];
        const double f3 = gg.grid_ptr_cu[(i_local + 1) * gg.y_size + j_local];
        const double f4 = gg.grid_ptr_cu[(i_local + 1) * gg.y_size + j_local + 1];

        // std::cout << "i_local: " << i_local << " j_local: " << j_local << std::endl;
        // std::cout << "f1: " << f1 << " f2: " << f2 << " f3: " << f3 << " f4: " << f4
        //           << std::endl;

        // std::cout << std::endl;

        // check if qs is correct
        // if (x_Q < x_d_min || x_Q > x_d_max || y_Q < y_d_min || y_Q > y_d_max) {
        //     std::cout << "x_Q: " << x_Q << " y_Q: " << y_Q << std::endl;
        //     std::cout << "x_d_min: " << x_d_min << " x_d_max: " << x_d_max << std::endl;
        //     std::cout << "y_d_min: " << y_d_min << " y_d_max: " << y_d_max << std::endl;
        //     return false;
        // }

        const double x1 = gg.x_zero + i_local * gg.delta;        // 2 * qr_size * ds_nr_domains
        const double x2 = gg.x_zero + (i_local + 1) * gg.delta;  // 3 * qr_size * ds_nr_domains
        const double y1 = gg.y_zero + j_local * gg.delta;        // 1 * qr_size * ds_nr_domains
        const double y2 = gg.y_zero + (j_local + 1) * gg.delta;  // 3 * qr_size * ds_nr_domains

        // std::cout << "x1: " << x1 << " x2: " << x2 << " y1: " << y1 << " y2: " << y2
        //           << std::endl;

        // 5 * 4 * qr_size * ds.nr_domains
        const double w11 = (x2 - x_Q) * (y2 - y_Q) / (gg.delta * gg.delta);
        const double w12 = (x2 - x_Q) * (y_Q - y1) / (gg.delta * gg.delta);
        const double w21 = (x_Q - x1) * (y2 - y_Q) / (gg.delta * gg.delta);
        const double w22 = (x_Q - x1) * (y_Q - y1) / (gg.delta * gg.delta);

        // std::cout << "w11: " << w11 << " w12: " << w12 << " w21: " << w21 << " w22: " << w22
        //           << std::endl;

        // 7 * qr_size * ds.nr_domains
        const double f_Q = w11 * f1 + w12 * f2 + w21 * f3 + w22 * f4;

        // 3 * qr_size * ds.nr_domains
        // data transfer 8 * qr_size * ds.nr_domains
        Qs_i += f_Q * qr.weights_ptr_cu[q_i] * volume;

        // std::cout << "---gg" << std::endl;
    }

    // Qs[i] = Qs_i;
    // }

    Qs[domain_nr] = Qs_i;

    return;
}

/**
 * @brief Kernel function to perform the quadrature in the global domain.
 *
 * @param Qs
 * @param gg
 * @param qr
 * @param qr_nodes_nr_
 * @return __global__
 */
__global__ void perform_quadrature_global_stripe_kernel(double* Qs,                  //
                                                        const global_grid_type& gg,  //
                                                        const quadrature_rule& qr,   //
                                                        const size_t qr_nodes_nr_,   //
                                                        const domains_stripe& ds) {  //
    //
    const size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t domain_nr = threadIdx.x;
    const size_t stripe_nr = blockIdx.x;
}

/**
 * @brief
 *
 * @param gg
 * @return true
 * @return false
 */
bool copy_global_grid_to_device(const global_grid_type& gg) {
    hipError_t e1 = hipMalloc((void**)&gg.grid_ptr_cu,  //
                              (unsigned long)gg.grid.size() * sizeof(double));

    hipError_t e2 = hipMemcpy(gg.grid_ptr_cu,
                              &gg.grid[0],
                              (unsigned long)gg.grid.size() * sizeof(double),
                              hipMemcpyHostToDevice);

    if (e1 != hipSuccess || e2 != hipSuccess) {
        return false;
    }

    return true;
}

/**
 * @brief
 *
 * @param gg
 * @return true
 * @return false
 */
bool free_global_grid_on_device(global_grid_type& gg) {
    hipError_t e1 = hipFree(gg.grid_ptr_cu);

    if (e1 != hipSuccess) {
        return false;
    }

    gg.grid_ptr_cu = nullptr;

    return true;
}

/**
 * @brief Copies the global grid to the device.
 *
 * @param qr
 * @return true
 * @return false
 */
bool copy_quadrature_rule_to_device(const quadrature_rule& qr) {
    hipError_t e1 = hipMalloc((void**)&qr.x_nodes_ptr_cu,  //
                              (unsigned long)qr.x_nodes.size() * sizeof(double));

    hipError_t e2 = hipMalloc((void**)&qr.y_nodes_ptr_cu,  //
                              (unsigned long)qr.y_nodes.size() * sizeof(double));

    hipError_t e3 = hipMalloc((void**)&qr.weights_ptr_cu,  //
                              (unsigned long)qr.weights.size() * sizeof(double));

    hipError_t e4 = hipMemcpy(qr.x_nodes_ptr_cu,
                              &qr.x_nodes[0],
                              (unsigned long)qr.x_nodes.size() * sizeof(double),
                              hipMemcpyHostToDevice);

    hipError_t e5 = hipMemcpy(qr.y_nodes_ptr_cu,
                              &qr.y_nodes[0],
                              (unsigned long)qr.y_nodes.size() * sizeof(double),
                              hipMemcpyHostToDevice);

    hipError_t e6 = hipMemcpy(qr.weights_ptr_cu,
                              &qr.weights[0],
                              (unsigned long)qr.weights.size() * sizeof(double),
                              hipMemcpyHostToDevice);

    if (e1 != hipSuccess || e2 != hipSuccess || e3 != hipSuccess || e4 != hipSuccess ||
        e5 != hipSuccess || e6 != hipSuccess) {
        return false;
    }

    return true;
}

/**
 * @brief Frees the quadrature rule on the device.
 *
 * @param qr
 * @return true
 * @return false
 */
bool free_quadrature_rule_on_device(quadrature_rule& qr) {
    hipError_t e1 = hipFree(qr.x_nodes_ptr_cu);
    hipError_t e2 = hipFree(qr.y_nodes_ptr_cu);
    hipError_t e3 = hipFree(qr.weights_ptr_cu);

    if (e1 != hipSuccess || e2 != hipSuccess || e3 != hipSuccess) {
        return false;
    }

    qr.x_nodes_ptr_cu = nullptr;
    qr.y_nodes_ptr_cu = nullptr;
    qr.weights_ptr_cu = nullptr;

    return true;
}

/**
 * @brief
 *
 */
extern "C" int test_grid_cuda() {
    int dev;
    hipGetDevice(&dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\n");

    printf("Device:                %s\n", deviceProp.name);
    printf("CUDA Capability:       %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Memory available:      %.3lf Gbytes\n",
           (double)deviceProp.totalGlobalMem / (double)(1024 * 1024 * 1024));
    printf("Number of SMs:         %d\n", deviceProp.multiProcessorCount);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max threads per SM:    %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Shared memory per SM:  %d\n", deviceProp.sharedMemPerMultiprocessor);
    printf("Number of SP:          %d\n", getSPcores(deviceProp));
    printf("Warp size:             %d\n", deviceProp.warpSize);
    printf("Max lane per SM:       %d\n", getSPcores(deviceProp) / deviceProp.multiProcessorCount);

    printf("\n");

    move_global_grid_to_device(gg);

    free_global_grid_on_device(gg);

    return 0;
}