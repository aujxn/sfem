
extern "C" {
#include "sfem_base.h"
}

#include "sfem_cuda_base.h"


extern "C" void crs_device_create(const ptrdiff_t nnodes,
                                  const ptrdiff_t nnz,
                                  count_t** rowptr,
                                  idx_t** colidx,
                                  real_t** values)

{  
    SFEM_CUDA_CHECK(hipMalloc(rowptr, (nnodes + 1) * sizeof(count_t)));
    SFEM_CUDA_CHECK(hipMalloc(colidx, nnz * sizeof(idx_t)));
    SFEM_CUDA_CHECK(hipMalloc(values, nnz * sizeof(real_t)));
}

extern "C" void crs_graph_device_create(const ptrdiff_t nnodes,
                                  const ptrdiff_t nnz,
                                  count_t** rowptr,
                                  idx_t** colidx)

{  
    SFEM_CUDA_CHECK(hipMalloc(rowptr, (nnodes + 1) * sizeof(count_t)));
    SFEM_CUDA_CHECK(hipMalloc(colidx, nnz * sizeof(idx_t)));
}

extern "C" void crs_device_free(count_t* rowptr, idx_t* colidx, real_t* values) {
    SFEM_CUDA_CHECK(hipFree(rowptr));
    SFEM_CUDA_CHECK(hipFree(colidx));
    SFEM_CUDA_CHECK(hipFree(values));
}

extern "C" void crs_graph_device_free(count_t* rowptr, idx_t* colidx) {
    SFEM_CUDA_CHECK(hipFree(rowptr));
    SFEM_CUDA_CHECK(hipFree(colidx));
}

extern "C" void crs_graph_host_to_device(const ptrdiff_t nnodes,
                                         const ptrdiff_t nnz,
                                         const count_t* const SFEM_RESTRICT h_rowptr,
                                         const idx_t* const SFEM_RESTRICT h_colidx,
                                         count_t* const SFEM_RESTRICT d_rowptr,
                                         idx_t* const SFEM_RESTRICT d_colidx) {
    SFEM_CUDA_CHECK(hipMemcpy(d_rowptr, h_rowptr, (nnodes + 1) * sizeof(count_t), hipMemcpyHostToDevice));
    SFEM_CUDA_CHECK(hipMemcpy(d_colidx, h_colidx, nnz * sizeof(idx_t), hipMemcpyHostToDevice));
}
