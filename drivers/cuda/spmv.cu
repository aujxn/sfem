
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>  // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>

// https://docs.nvidia.com/cuda/cusparse/index.html#compressed-sparse-row-format-csr

#define CHECK_CUDA(func)                                               \
    do {                                                               \
        hipError_t status = (func);                                   \
        if (status != hipSuccess) {                                   \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__,                                           \
                   hipGetErrorString(status),                         \
                   status);                                            \
            return EXIT_FAILURE;                                       \
        }                                                              \
    } while (0)

#define CHECK_CUSPARSE(func)                                               \
    do {                                                                   \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                           \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__,                                               \
                   hipsparseGetErrorString(status),                         \
                   status);                                                \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    } while (0)

// make spmv cuda=1
int main() {
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    double alpha = 1, beta = 0;

    hipsparseDnVecDescr_t vecX, vecY;
    void *dX, *dY;

    hipsparseSpMatDescr_t d_matrix;
    int64_t rows;
    int64_t cols;
    int64_t nnz;
    void* csrRowOffsets;
    void* csrColInd;
    void* csrValues;

    hipsparseIndexType_t csrRowOffsetsType = HIPSPARSE_INDEX_32I;
    hipsparseIndexType_t csrColIndType = HIPSPARSE_INDEX_32I;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
    hipDataType valueType = HIP_R_64F;

    CHECK_CUSPARSE(hipsparseCreateCsr(&d_matrix,
                                     rows,
                                     cols,
                                     nnz,
                                     csrRowOffsets,
                                     csrColInd,
                                     csrValues,
                                     csrRowOffsetsType,
                                     csrColIndType,
                                     idxBase,
                                     valueType));

    // Create dense vectors
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, cols, dX, valueType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, rows, dY, valueType));

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           d_matrix,
                                           vecX,
                                           &beta,
                                           vecY,
                                           valueType,
                                           HIPSPARSE_SPMV_ALG_DEFAULT,
                                           &bufferSize));

    void* dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    CHECK_CUSPARSE(hipsparseSpMV(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                d_matrix,
                                vecX,
                                &beta,
                                vecY,
                                valueType,
                                HIPSPARSE_SPMV_ALG_DEFAULT,
                                dBuffer));

    hipDeviceSynchronize();

    CHECK_CUSPARSE(hipsparseDestroySpMat(d_matrix));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUDA(hipFree(csrRowOffsets));
    CHECK_CUDA(hipFree(csrColInd));
    CHECK_CUDA(hipFree(csrValues));
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dY));
    return 0;
}