#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
// #include <cstdio>
#include <algorithm>
#include <cstddef>

extern "C" {
#include "sfem_base.h"
#include "sfem_vec.h"
#include "sortreduce.h"
}

#include "sfem_cuda_base.h"
#include "sfem_mesh.h"

#include "macro_tet4_laplacian_incore_cuda.h"

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define POW2(a) ((a) * (a))

static inline __device__ __host__ void fff_micro_kernel(const geom_t px0,
                                                        const geom_t px1,
                                                        const geom_t px2,
                                                        const geom_t px3,
                                                        const geom_t py0,
                                                        const geom_t py1,
                                                        const geom_t py2,
                                                        const geom_t py3,
                                                        const geom_t pz0,
                                                        const geom_t pz1,
                                                        const geom_t pz2,
                                                        const geom_t pz3,
                                                        const count_t stride,
                                                        geom_t *fff) {
    const geom_t x0 = -px0 + px1;
    const geom_t x1 = -py0 + py2;
    const geom_t x2 = -pz0 + pz3;
    const geom_t x3 = x1 * x2;
    const geom_t x4 = -pz0 + pz1;
    const geom_t x5 = -px0 + px2;
    const geom_t x6 = -py0 + py3;
    const geom_t x7 = x5 * x6;
    const geom_t x8 = -py0 + py1;
    const geom_t x9 = -px0 + px3;
    const geom_t x10 = -pz0 + pz2;
    const geom_t x11 = x10 * x6;
    const geom_t x12 = x2 * x5;
    const geom_t x13 = x1 * x9;
    const geom_t x14 = -x0 * x11 + x0 * x3 + x10 * x8 * x9 - x12 * x8 - x13 * x4 + x4 * x7;
    const geom_t x15 = -x13 + x7;
    const geom_t x16 = 1 / POW2(x14);
    const geom_t x17 = x10 * x9 - x12;
    const geom_t x18 = -x11 + x3;
    const geom_t x19 = -x0 * x6 + x8 * x9;
    const geom_t x20 = x15 * x16;
    const geom_t x21 = x0 * x2 - x4 * x9;
    const geom_t x22 = x16 * x17;
    const geom_t x23 = -x2 * x8 + x4 * x6;
    const geom_t x24 = x16 * x18;
    const geom_t x25 = x0 * x1 - x5 * x8;
    const geom_t x26 = -x0 * x10 + x4 * x5;
    const geom_t x27 = -x1 * x4 + x10 * x8;
    fff[0 * stride] = x14 * (POW2(x15) * x16 + x16 * POW2(x17) + x16 * POW2(x18));
    fff[1 * stride] = x14 * (x19 * x20 + x21 * x22 + x23 * x24);
    fff[2 * stride] = x14 * (x20 * x25 + x22 * x26 + x24 * x27);
    fff[3 * stride] = x14 * (x16 * POW2(x19) + x16 * POW2(x21) + x16 * POW2(x23));
    fff[4 * stride] = x14 * (x16 * x19 * x25 + x16 * x21 * x26 + x16 * x23 * x27);
    fff[5 * stride] = x14 * (x16 * POW2(x25) + x16 * POW2(x26) + x16 * POW2(x27));
}

static /*inline*/ __device__ __host__ void sub_fff_0(const geom_t *const SFEM_RESTRICT fff,
                                                     const ptrdiff_t stride,
                                                     geom_t *const SFEM_RESTRICT sub_fff) {
    sub_fff[0] = (geom_t)(1.0 / 2.0) * fff[0 * stride];
    sub_fff[1] = (geom_t)(1.0 / 2.0) * fff[1 * stride];
    sub_fff[2] = (geom_t)(1.0 / 2.0) * fff[2 * stride];
    sub_fff[3] = (geom_t)(1.0 / 2.0) * fff[3 * stride];
    sub_fff[4] = (geom_t)(1.0 / 2.0) * fff[4 * stride];
    sub_fff[5] = (geom_t)(1.0 / 2.0) * fff[5 * stride];
}

static /*inline*/ __device__ __host__ void sub_fff_4(const geom_t *const SFEM_RESTRICT fff,
                                                     const ptrdiff_t stride,
                                                     geom_t *const SFEM_RESTRICT sub_fff) {
    const geom_t x0 = (geom_t)(1.0 / 2.0) * fff[0 * stride];
    const geom_t x1 = (geom_t)(1.0 / 2.0) * fff[2 * stride];
    sub_fff[0] = fff[1 * stride] + (1.0 / 2.0) * fff[3 * stride] + x0;
    sub_fff[1] = (geom_t)(-1.0 / 2.0) * fff[1 * stride] - x0;
    sub_fff[2] = (geom_t)(1.0 / 2.0) * fff[4 * stride] + x1;
    sub_fff[3] = x0;
    sub_fff[4] = -x1;
    sub_fff[5] = (geom_t)(1.0 / 2.0) * fff[5 * stride];
}

static /*inline*/ __device__ __host__ void sub_fff_5(const geom_t *const SFEM_RESTRICT fff,
                                                     const ptrdiff_t stride,
                                                     geom_t *const SFEM_RESTRICT sub_fff) {
    const geom_t x0 = (geom_t)(1.0 / 2.0) * fff[3 * stride];
    const geom_t x1 = fff[4 * stride] + (geom_t)(1.0 / 2.0) * fff[5 * stride] + x0;
    const geom_t x2 = (geom_t)(1.0 / 2.0) * fff[4 * stride] + x0;
    const geom_t x3 = (geom_t)(1.0 / 2.0) * fff[1 * stride];
    sub_fff[0] = x1;
    sub_fff[1] = -x2;
    sub_fff[2] = (geom_t)(-1.0 / 2.0) * fff[2 * stride] - x1 - x3;
    sub_fff[3] = x0;
    sub_fff[4] = x2 + x3;
    sub_fff[5] = (geom_t)(1.0 / 2.0) * fff[0 * stride] + fff[1 * stride] + fff[2 * stride] + x1;
}

static /*inline*/ __device__ __host__ void sub_fff_6(const geom_t *const SFEM_RESTRICT fff,
                                                     const ptrdiff_t stride,
                                                     geom_t *const SFEM_RESTRICT sub_fff) {
    const geom_t x0 = (geom_t)(1.0 / 2.0) * fff[3 * stride];
    const geom_t x1 = (geom_t)(1.0 / 2.0) * fff[4 * stride];
    const geom_t x2 = (geom_t)(1.0 / 2.0) * fff[1 * stride] + x0;
    sub_fff[0] = (geom_t)(1.0 / 2.0) * fff[0 * stride] + fff[1 * stride] + x0;
    sub_fff[1] = (geom_t)(1.0 / 2.0) * fff[2 * stride] + x1 + x2;
    sub_fff[2] = -x2;
    sub_fff[3] = fff[4 * stride] + (geom_t)(1.0 / 2.0) * fff[5 * stride] + x0;
    sub_fff[4] = -x0 - x1;
    sub_fff[5] = x0;
}

static /*inline*/ __device__ __host__ void sub_fff_7(const geom_t *const SFEM_RESTRICT fff,
                                                     const ptrdiff_t stride,
                                                     geom_t *const SFEM_RESTRICT sub_fff) {
    const geom_t x0 = (geom_t)(1.0 / 2.0) * fff[5 * stride];
    const geom_t x1 = (geom_t)(1.0 / 2.0) * fff[2 * stride];
    sub_fff[0] = x0;
    sub_fff[1] = (geom_t)(-1.0 / 2.0) * fff[4 * stride] - x0;
    sub_fff[2] = -x1;
    sub_fff[3] = (geom_t)(1.0 / 2.0) * fff[3 * stride] + fff[4 * stride] + x0;
    sub_fff[4] = (geom_t)(1.0 / 2.0) * fff[1 * stride] + x1;
    sub_fff[5] = (geom_t)(1.0 / 2.0) * fff[0 * stride];
}

static /*inline*/ __device__ __host__ void lapl_apply_micro_kernel(const geom_t *const SFEM_RESTRICT
                                                                       fff,
                                                                   const real_t u0,
                                                                   const real_t u1,
                                                                   const real_t u2,
                                                                   const real_t u3,
                                                                   real_t *const SFEM_RESTRICT e0,
                                                                   real_t *const SFEM_RESTRICT e1,
                                                                   real_t *const SFEM_RESTRICT e2,
                                                                   real_t *const SFEM_RESTRICT e3) {
    const real_t x0 = (1.0 / 6.0) * u0;
    const real_t x1 = fff[0] * x0;
    const real_t x2 = (1.0 / 6.0) * u1;
    const real_t x3 = fff[0] * x2;
    const real_t x4 = fff[1] * x2;
    const real_t x5 = (1.0 / 6.0) * u2;
    const real_t x6 = fff[1] * x5;
    const real_t x7 = fff[2] * x2;
    const real_t x8 = (1.0 / 6.0) * u3;
    const real_t x9 = fff[2] * x8;
    const real_t x10 = fff[3] * x0;
    const real_t x11 = fff[3] * x5;
    const real_t x12 = fff[4] * x5;
    const real_t x13 = fff[4] * x8;
    const real_t x14 = fff[5] * x0;
    const real_t x15 = fff[5] * x8;
    const real_t x16 = fff[1] * x0;
    const real_t x17 = fff[2] * x0;
    const real_t x18 = fff[4] * x0;
    *e0 += (1.0 / 3.0) * fff[1] * u0 + (1.0 / 3.0) * fff[2] * u0 + (1.0 / 3.0) * fff[4] * u0 + x1 +
           x10 - x11 - x12 - x13 + x14 - x15 - x3 - x4 - x6 - x7 - x9;
    *e1 += -x1 - x16 - x17 + x3 + x6 + x9;
    *e2 += -x10 + x11 + x13 - x16 - x18 + x4;
    *e3 += x12 - x14 + x15 - x17 - x18 + x7;
}

__global__ void macro_tet4_cuda_incore_laplacian_apply_kernel(const ptrdiff_t nelements,
                                                              idx_t *const SFEM_RESTRICT elems,
                                                              const geom_t *const SFEM_RESTRICT fff,
                                                              const real_t *const SFEM_RESTRICT x,
                                                              real_t *const SFEM_RESTRICT y) {
    real_t ex[10];
    real_t ey[10];
    geom_t sub_fff[6];

    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            ey[v] = 0;
        }
        // collect coeffs
#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            ex[v] = x[elems[v * nelements + e]];
        }

        // apply operator

        {  // Corner tests
            sub_fff_0(&fff[e], nelements, sub_fff);

            // [0, 4, 6, 7],
            lapl_apply_micro_kernel(sub_fff,
                                    ex[0],
                                    ex[4],
                                    ex[6],
                                    ex[7],  //
                                    &ey[0],
                                    &ey[4],
                                    &ey[6],
                                    &ey[7]);

            // [4, 1, 5, 8],
            lapl_apply_micro_kernel(sub_fff,
                                    ex[4],
                                    ex[1],
                                    ex[5],
                                    ex[8],  //
                                    &ey[4],
                                    &ey[1],
                                    &ey[5],
                                    &ey[8]);

            // [6, 5, 2, 9],
            lapl_apply_micro_kernel(sub_fff,
                                    ex[6],
                                    ex[5],
                                    ex[2],
                                    ex[9],  //
                                    &ey[6],
                                    &ey[5],
                                    &ey[2],
                                    &ey[9]);

            // [7, 8, 9, 3],
            lapl_apply_micro_kernel(sub_fff,
                                    ex[7],
                                    ex[8],
                                    ex[9],
                                    ex[3],  //
                                    &ey[7],
                                    &ey[8],
                                    &ey[9],
                                    &ey[3]);
        }

        {  // Octahedron tets

            // [4, 5, 6, 8],
            sub_fff_4(&fff[e], nelements, sub_fff);
            lapl_apply_micro_kernel(sub_fff,
                                    ex[4],
                                    ex[5],
                                    ex[6],
                                    ex[8],  //
                                    &ey[4],
                                    &ey[5],
                                    &ey[6],
                                    &ey[8]);

            // [7, 4, 6, 8],
            sub_fff_5(&fff[e], nelements, sub_fff);
            lapl_apply_micro_kernel(sub_fff,
                                    ex[7],
                                    ex[4],
                                    ex[6],
                                    ex[8],  //
                                    &ey[7],
                                    &ey[4],
                                    &ey[6],
                                    &ey[8]);

            // [6, 5, 9, 8],
            sub_fff_6(&fff[e], nelements, sub_fff);
            lapl_apply_micro_kernel(sub_fff,
                                    ex[6],
                                    ex[5],
                                    ex[9],
                                    ex[8],  //
                                    &ey[6],
                                    &ey[5],
                                    &ey[9],
                                    &ey[8]);

            // [7, 6, 9, 8]]
            sub_fff_7(&fff[e], nelements, sub_fff);
            lapl_apply_micro_kernel(sub_fff,
                                    ex[7],
                                    ex[6],
                                    ex[9],
                                    ex[8],  //
                                    &ey[7],
                                    &ey[6],
                                    &ey[9],
                                    &ey[8]);
        }

        // redistribute coeffs
#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            atomicAdd(&y[elems[v * nelements + e]], ey[v]);
        }
    }
}

extern int macro_tet4_cuda_incore_laplacian_apply(cuda_incore_laplacian_t *ctx,
                                                  const real_t *const d_x,
                                                  real_t *const d_y) {
    static int block_size = 128;
    ptrdiff_t n_blocks = std::max(ptrdiff_t(1), (ctx->nelements + block_size - 1) / block_size);
    macro_tet4_cuda_incore_laplacian_apply_kernel<<<n_blocks, block_size, 0>>>(
        ctx->nelements, ctx->d_elems, ctx->d_fff, d_x, d_y);
    return 0;
}

extern int macro_tet4_cuda_incore_laplacian_init(cuda_incore_laplacian_t *ctx, mesh_t mesh) {
    {  // Create FFF and store it on device
        geom_t *h_fff = (geom_t *)calloc(6 * mesh.nelements, sizeof(geom_t));

#pragma omp parallel
        {
#pragma omp for
            for (ptrdiff_t e = 0; e < mesh.nelements; e++) {
                fff_micro_kernel(mesh.points[0][mesh.elements[0][e]],
                                 mesh.points[0][mesh.elements[1][e]],
                                 mesh.points[0][mesh.elements[2][e]],
                                 mesh.points[0][mesh.elements[3][e]],
                                 mesh.points[1][mesh.elements[0][e]],
                                 mesh.points[1][mesh.elements[1][e]],
                                 mesh.points[1][mesh.elements[2][e]],
                                 mesh.points[1][mesh.elements[3][e]],
                                 mesh.points[2][mesh.elements[0][e]],
                                 mesh.points[2][mesh.elements[1][e]],
                                 mesh.points[2][mesh.elements[2][e]],
                                 mesh.points[2][mesh.elements[3][e]],
                                 mesh.nelements,
                                 &h_fff[e]);
            }
        }

        SFEM_CUDA_CHECK(hipMalloc(&ctx->d_fff, 6 * mesh.nelements * sizeof(geom_t)));
        SFEM_CUDA_CHECK(hipMemcpy(
            ctx->d_fff, h_fff, 6 * mesh.nelements * sizeof(geom_t), hipMemcpyHostToDevice));
        free(h_fff);
    }

    {
        // Store elem indices on device
        SFEM_CUDA_CHECK(hipMalloc(&ctx->d_elems, 10 * mesh.nelements * sizeof(idx_t)));

        for (int d = 0; d < 10; d++) {
            SFEM_CUDA_CHECK(hipMemcpy(ctx->d_elems + d * mesh.nelements,
                                       mesh.elements[d],
                                       mesh.nelements * sizeof(idx_t),
                                       hipMemcpyHostToDevice));
        }
    }

    ctx->nelements = mesh.nelements;
    return 0;
}

extern int macro_tet4_cuda_incore_laplacian_destroy(cuda_incore_laplacian_t *ctx) {
    hipFree(ctx->d_elems);
    hipFree(ctx->d_fff);

    ctx->nelements = 0;
    ctx->d_elems = nullptr;
    ctx->d_fff = nullptr;
    return 0;
}
