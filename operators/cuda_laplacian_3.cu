#include "hip/hip_runtime.h"
// #include "laplacian.h"

#include <cassert>
#include <cmath>
// #include <cstdio>
#include <algorithm>
#include <cstddef>

extern "C" {
#include "sfem_base.h"

#include "crs_graph.h"
#include "cuda_crs.h"
#include "sfem_base.h"
#include "sfem_vec.h"
#include "sortreduce.h"
}

#include "sfem_cuda_base.h"

#if 1
#include "nvToolsExt.h"
#define SFEM_RANGE_PUSH(name_) \
    do {                       \
        nvtxRangePushA(name_); \
    } while (0)
#define SFEM_RANGE_POP() \
    do {                 \
        nvtxRangePop();  \
    } while (0)
#else
#define SFEM_RANGE_PUSH(name_)
#define SFEM_RANGE_POP()
#endif

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define POW2(a) ((a) * (a))

static inline __device__ void laplacian(const real_t *SFEM_RESTRICT jac_inv,
                                        const count_t stride,
                                        real_t *SFEM_RESTRICT element_matrix)

{
    // FLOATING POINT OPS!
    //      - Result: ADD + 16*ASSIGNMENT + 3*MUL
    //      - Subexpressions: 3*NEG + 6*SUB
    const real_t x0 = -jac_inv[0 * stride] - jac_inv[1 * stride] - jac_inv[2 * stride];
    const real_t x1 = -jac_inv[1 * stride] - jac_inv[3 * stride] - jac_inv[4 * stride];
    const real_t x2 = -jac_inv[2 * stride] - jac_inv[4 * stride] - jac_inv[5 * stride];
    element_matrix[0 * stride] = jac_inv[0 * stride] + 2 * jac_inv[1 * stride] + 2 * jac_inv[2 * stride] +
                                 jac_inv[3 * stride] + 2 * jac_inv[4 * stride] + jac_inv[5 * stride];

    element_matrix[1 * stride] = x0;
    element_matrix[2 * stride] = x1;
    element_matrix[3 * stride] = x2;
    element_matrix[4 * stride] = x0;
    element_matrix[5 * stride] = jac_inv[0 * stride];
    element_matrix[6 * stride] = jac_inv[1 * stride];
    element_matrix[7 * stride] = jac_inv[2 * stride];
    element_matrix[8 * stride] = x1;
    element_matrix[9 * stride] = jac_inv[1 * stride];
    element_matrix[10 * stride] = jac_inv[3 * stride];
    element_matrix[11 * stride] = jac_inv[4 * stride];
    element_matrix[12 * stride] = x2;
    element_matrix[13 * stride] = jac_inv[2 * stride];
    element_matrix[14 * stride] = jac_inv[4 * stride];
    element_matrix[15 * stride] = jac_inv[5 * stride];
}

static inline __device__ __host__ int linear_search(const idx_t target, const idx_t *const arr, const int size) {
    int i;
    for (i = 0; i < size - SFEM_VECTOR_SIZE; i += SFEM_VECTOR_SIZE) {
        if (arr[i] == target) return i;
        if (arr[i + 1] == target) return i + 1;
        if (arr[i + 2] == target) return i + 2;
        if (arr[i + 3] == target) return i + 3;
    }
    for (; i < size; i++) {
        if (arr[i] == target) return i;
    }
    return -1;
}

static inline __device__ __host__ int find_col(const idx_t key, const idx_t *const row, const int lenrow) {
    // if (lenrow <= 32)
    // {
    return linear_search(key, row, lenrow);

    // Using sentinel (potentially dangerous if matrix is buggy and column does not exist)
    // while (key > row[++k]) {
    //     // Hi
    // }
    // assert(k < lenrow);
    // assert(key == row[k]);
    // } else {
    //     // Use this for larger number of dofs per row
    //     return find_idx_binary_search(key, row, lenrow);
    // }
}

static inline __device__ __host__ void find_cols4(const idx_t *targets,
                                                  const idx_t *const row,
                                                  const int lenrow,
                                                  int *ks) {
    if (lenrow > 32) {
        for (int d = 0; d < 4; ++d) {
            ks[d] = find_col(targets[d], row, lenrow);
        }
    } else {
#pragma unroll(4)
        for (int d = 0; d < 4; ++d) {
            ks[d] = 0;
        }

        for (int i = 0; i < lenrow; ++i) {
#pragma unroll(4)
            for (int d = 0; d < 4; ++d) {
                ks[d] += row[i] < targets[d];
            }
        }
    }
}

static inline __device__ __host__ void fff_micro_kernel(const real_t px0,
                                                        const real_t px1,
                                                        const real_t px2,
                                                        const real_t px3,
                                                        const real_t py0,
                                                        const real_t py1,
                                                        const real_t py2,
                                                        const real_t py3,
                                                        const real_t pz0,
                                                        const real_t pz1,
                                                        const real_t pz2,
                                                        const real_t pz3,
                                                        const count_t stride,
                                                        real_t *jac_inv) {
    //      - Result: 6*ADD + 6*ASSIGNMENT + 24*MUL + 9*POW
    //      - Subexpressions: 4*ADD + 6*DIV + 28*MUL + NEG + POW + 24*SUB
    const real_t x0 = -px0 + px1;
    const real_t x1 = -py0 + py2;
    const real_t x2 = -pz0 + pz3;
    const real_t x3 = x1 * x2;
    const real_t x4 = x0 * x3;
    const real_t x5 = -py0 + py3;
    const real_t x6 = -pz0 + pz2;
    const real_t x7 = x5 * x6;
    const real_t x8 = x0 * x7;
    const real_t x9 = -py0 + py1;
    const real_t x10 = -px0 + px2;
    const real_t x11 = x10 * x2;
    const real_t x12 = x11 * x9;
    const real_t x13 = -pz0 + pz1;
    const real_t x14 = x10 * x5;
    const real_t x15 = x13 * x14;
    const real_t x16 = -px0 + px3;
    const real_t x17 = x16 * x6 * x9;
    const real_t x18 = x1 * x16;
    const real_t x19 = x13 * x18;
    const real_t x20 =
        -1.0 / 6.0 * x12 + (1.0 / 6.0) * x15 + (1.0 / 6.0) * x17 - 1.0 / 6.0 * x19 + (1.0 / 6.0) * x4 - 1.0 / 6.0 * x8;
    const real_t x21 = x14 - x18;
    const real_t x22 = 1./POW2(-x12 + x15 + x17 - x19 + x4 - x8);
    const real_t x23 = -x11 + x16 * x6;
    const real_t x24 = x3 - x7;
    const real_t x25 = -x0 * x5 + x16 * x9;
    const real_t x26 = x21 * x22;
    const real_t x27 = x0 * x2 - x13 * x16;
    const real_t x28 = x22 * x23;
    const real_t x29 = x13 * x5 - x2 * x9;
    const real_t x30 = x22 * x24;
    const real_t x31 = x0 * x1 - x10 * x9;
    const real_t x32 = -x0 * x6 + x10 * x13;
    const real_t x33 = -x1 * x13 + x6 * x9;
    jac_inv[0 * stride] = x20 * (POW2(x21) * x22 + x22 * POW2(x23) + x22 * POW2(x24));
    jac_inv[1 * stride] = x20 * (x25 * x26 + x27 * x28 + x29 * x30);
    jac_inv[2 * stride] = x20 * (x26 * x31 + x28 * x32 + x30 * x33);
    jac_inv[3 * stride] = x20 * (x22 * POW2(x25) + x22 * POW2(x27) + x22 * POW2(x29));
    jac_inv[4 * stride] = x20 * (x22 * x25 * x31 + x22 * x27 * x32 + x22 * x29 * x33);
    jac_inv[5 * stride] = x20 * (x22 * POW2(x31) + x22 * POW2(x32) + x22 * POW2(x33));
}

__global__ void fff_kernel(const ptrdiff_t nelements,
                                        const geom_t *const SFEM_RESTRICT xyz,
                                        real_t *const SFEM_RESTRICT fff) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
        // Thy element coordinates and jacobian
        const geom_t *const this_xyz = &xyz[e];
        real_t *const this_fff = &fff[e];

        const ptrdiff_t xi = 0 * 4;
        const ptrdiff_t yi = 1 * 4;
        const ptrdiff_t zi = 2 * 4;

        fff_micro_kernel(
            // X-coordinates
            this_xyz[(xi + 0) * nelements],
            this_xyz[(xi + 1) * nelements],
            this_xyz[(xi + 2) * nelements],
            this_xyz[(xi + 3) * nelements],
            // Y-coordinates
            this_xyz[(yi + 0) * nelements],
            this_xyz[(yi + 1) * nelements],
            this_xyz[(yi + 2) * nelements],
            this_xyz[(yi + 3) * nelements],
            // Z-coordinates
            this_xyz[(zi + 0) * nelements],
            this_xyz[(zi + 1) * nelements],
            this_xyz[(zi + 2) * nelements],
            this_xyz[(zi + 3) * nelements],
            nelements,
            this_fff);
    }
}

__global__ void laplacian_assemble_hessian_kernel(const ptrdiff_t nelements,
                                                  const real_t *const SFEM_RESTRICT fff,
                                                  real_t *const SFEM_RESTRICT values) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
        laplacian(&fff[e], nelements, &values[e]);
    }
}

__global__ void local_to_global_kernel(const ptrdiff_t nelements,
                                       idx_t **const SFEM_RESTRICT elems,
                                       const real_t *const SFEM_RESTRICT element_matrix,
                                       const count_t *const SFEM_RESTRICT rowptr,
                                       const idx_t *const SFEM_RESTRICT colidx,
                                       real_t *const SFEM_RESTRICT values) {
    idx_t ev[4];
    idx_t ks[4];
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements; e += blockDim.x * gridDim.x) {
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            ev[v] = elems[v][e];
        }

        // offsetted array for this element
        const real_t *const this_matrix = &element_matrix[e];

        // printf("%d)\n", (int)e);

        for (int edof_i = 0; edof_i < 4; ++edof_i) {
            const idx_t dof_i = ev[edof_i];
            const idx_t lenrow = rowptr[dof_i + 1] - rowptr[dof_i];

            const idx_t *const row = &colidx[rowptr[dof_i]];

            find_cols4(ev, row, lenrow, ks);

            real_t *const rowvalues = &values[rowptr[dof_i]];

            // #pragma unroll(4)
            for (int edof_j = 0; edof_j < 4; ++edof_j) {
                ptrdiff_t idx = (edof_i * 4 + edof_j) * nelements;
                const real_t v = this_matrix[idx];
                atomicAdd(&rowvalues[ks[edof_j]], v);
            }
        }
    }
}

__global__ void print_elem_kernel(const ptrdiff_t nelements, idx_t **const elems) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= nelements) return;

    printf("%d %d %d %d\n", elems[0][i], elems[1][i], elems[2][i], elems[3][i]);
}

extern "C" void laplacian_assemble_hessian(const ptrdiff_t nelements,
                                           const ptrdiff_t nnodes,
                                           idx_t **const SFEM_RESTRICT elems,
                                           geom_t **const SFEM_RESTRICT xyz,
                                           const count_t *const SFEM_RESTRICT rowptr,
                                           const idx_t *const SFEM_RESTRICT colidx,
                                           real_t *const SFEM_RESTRICT values) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    SFEM_RANGE_PUSH("lapl-set-up");
    hipEventRecord(start);

    // static int block_size = 256;
    static int block_size = 128;
    const ptrdiff_t nbatch = MIN(block_size * 500, nelements);

    ptrdiff_t n_blocks = std::max(ptrdiff_t(1), (nbatch + block_size - 1) / block_size);

    geom_t *he_xyz = nullptr;
    SFEM_CUDA_CHECK(hipHostMalloc(&he_xyz, 3 * 4 * nbatch * sizeof(geom_t)));
    geom_t *de_xyz = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_xyz, 3 * 4 * nbatch * sizeof(geom_t)));

    real_t *d_fff = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&d_fff, 6 * nbatch * sizeof(real_t)));

    real_t *he_matrix = nullptr;
    hipHostMalloc(&he_matrix, 4 * 4 * nbatch * sizeof(real_t));
    real_t *de_matrix = nullptr;
    SFEM_CUDA_CHECK(hipMalloc(&de_matrix, 4 * 4 * nbatch * sizeof(real_t)));

    idx_t *hh_elems[4];
    for (int d = 0; d < 4; d++) {
        SFEM_CUDA_CHECK(hipHostMalloc(&hh_elems[d], nbatch * sizeof(idx_t)));
    }

    idx_t **hd_elems[4];
    idx_t **d_elems = nullptr;

    count_t *d_rowptr = nullptr;
    idx_t *d_colidx = nullptr;
    real_t *d_values = nullptr;

    static const int nstreams = 4;
    hipStream_t stream[nstreams];
    hipEvent_t event[nstreams];
    for (int s = 0; s < nstreams; s++) {
        hipStreamCreate(&stream[s]);
        hipEventCreate(&event[s]);
    }

    // Allocate space for indices
    for (int d = 0; d < 4; d++) {
        SFEM_CUDA_CHECK(hipMalloc(&hd_elems[d], nbatch * sizeof(idx_t)));
    }

    SFEM_CUDA_CHECK(hipMalloc(&d_elems, 4 * sizeof(idx_t *)));
    hipMemcpy(d_elems, hd_elems, 4 * sizeof(idx_t *), hipMemcpyHostToDevice);

    SFEM_RANGE_POP();

    SFEM_RANGE_PUSH("lapl-crs-host-to-device");
    // Copy crs-matrix
    crs_device_create(nnodes, rowptr[nnodes], &d_rowptr, &d_colidx, &d_values);
    crs_graph_host_to_device(nnodes, rowptr[nnodes], rowptr, colidx, d_rowptr, d_colidx);

    SFEM_RANGE_POP();

    // TODO CRS HtoD async

    ptrdiff_t last_n = 0;
    ptrdiff_t last_element_offset = 0;
    for (ptrdiff_t element_offset = 0; element_offset < nelements; element_offset += nbatch) {
        ptrdiff_t n = MIN(nbatch, nelements - element_offset);


        /////////////////////////////////////////////////////////
        // Packing (stream 0)
        /////////////////////////////////////////////////////////

        if (last_n) {
            hipStreamSynchronize(stream[0]);
        }

        {
            SFEM_RANGE_PUSH("lapl-packing");
            {
                for (int d = 0; d < 3; ++d) {
                    for (int e_node = 0; e_node < 4; e_node++) {
                        const geom_t *const x = xyz[d];
                        ptrdiff_t offset = (d * 4 + e_node) * n;
                        const idx_t *const nodes = &elems[e_node][element_offset];

                        geom_t *buff = &he_xyz[offset];

#pragma omp parallel
                        {
#pragma omp for nowait
                            for (ptrdiff_t k = 0; k < n; k++) {
                                buff[k] = x[nodes[k]];
                            }
                        }
                    }
                }
            }

            SFEM_RANGE_POP();
        }


        /////////////////////////////////////////////////////////
        // Local to global (stream 3)
        /////////////////////////////////////////////////////////

        if (last_n) {
            // Make sure we have the elemental matrices and dof indices
            hipStreamWaitEvent(stream[3], event[1]);
            hipStreamWaitEvent(stream[3], event[2]);

            // Do this here to let the main kernel overlap with the packing
            local_to_global_kernel<<<n_blocks, block_size, 0, stream[3]>>>(
                last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);

            hipEventRecord(event[3], stream[3]);

            SFEM_DEBUG_SYNCHRONIZE();
        }

        /////////////////////////////////////////////////////////
        // XYZ HtoD (stream 0)
        /////////////////////////////////////////////////////////

        SFEM_CUDA_CHECK(hipMemcpyAsync(de_xyz, he_xyz, 3 * 4 * n * sizeof(geom_t), hipMemcpyHostToDevice, stream[0]));
        hipEventRecord(event[0], stream[0]);

        SFEM_DEBUG_SYNCHRONIZE();
        /////////////////////////////////////////////////////////
        // Jacobian computations (stream 1)
        /////////////////////////////////////////////////////////

        // Make sure we have the new XYZ coordinates
        hipStreamWaitEvent(stream[1], event[0]);

        fff_kernel<<<n_blocks, block_size, 0, stream[1]>>>(n, de_xyz, d_fff);

        SFEM_DEBUG_SYNCHRONIZE();
        /////////////////////////////////////////////////////////
        // DOF indices HtoD (stream 2)
        /////////////////////////////////////////////////////////

        // Ensure that previous HtoD is completed
        if(last_n) hipStreamSynchronize(stream[2]);

        SFEM_RANGE_PUSH("lapl-copy-host-to-host");
        //  Copy elements to host-pinned memory
        for (int e_node = 0; e_node < 4; e_node++) {
            memcpy(hh_elems[e_node], &elems[e_node][element_offset], n * sizeof(idx_t));
        }

        SFEM_RANGE_POP();

        // Make sure local to global has ended
        hipStreamWaitEvent(stream[2], event[3]);

        for (int e_node = 0; e_node < 4; e_node++) {
            SFEM_CUDA_CHECK(hipMemcpyAsync(
                hd_elems[e_node], hh_elems[e_node], n * sizeof(idx_t), hipMemcpyHostToDevice, stream[2]));
        }

        hipEventRecord(event[2], stream[2]);

        SFEM_DEBUG_SYNCHRONIZE();
        /////////////////////////////////////////////////////////
        // Assemble elemental matrices (stream 1)
        /////////////////////////////////////////////////////////

        // Make sure that we have new Jacobians
        hipStreamWaitEvent(stream[1], event[3]);

        laplacian_assemble_hessian_kernel<<<n_blocks, block_size, 0, stream[1]>>>(n, d_fff, de_matrix);
        hipEventRecord(event[1], stream[1]);

        SFEM_DEBUG_SYNCHRONIZE();
        /////////////////////////////////////////////////////////

        last_n = n;
        last_element_offset = element_offset;
    }

    /////////////////////////////////////////////////////////
    // Local to global (stream 3)
    /////////////////////////////////////////////////////////

    if (last_n) {

        // Make sure we have the elemental matrices and dof indices
        hipStreamWaitEvent(stream[3], event[1]);
        hipStreamWaitEvent(stream[3], event[2]);

        // Do this here to let the main kernel overlap with the packing
        local_to_global_kernel<<<n_blocks, block_size, 0, stream[3]>>>(
            last_n, d_elems, de_matrix, d_rowptr, d_colidx, d_values);

        SFEM_DEBUG_SYNCHRONIZE();

        hipStreamSynchronize(stream[3]);
    }

    /////////////////////////////////////////////////////////

    SFEM_RANGE_PUSH("lapl-values-device-to-host");

    SFEM_CUDA_CHECK(hipMemcpy(values, d_values, rowptr[nnodes] * sizeof(real_t), hipMemcpyDeviceToHost));

    SFEM_RANGE_POP();

    SFEM_RANGE_PUSH("lapl-tear-down");
    {  // Free resources on CPU
        hipHostFree(he_xyz);
        hipHostFree(he_matrix);

        for (int d = 0; d < 4; d++) {
            SFEM_CUDA_CHECK(hipHostFree(hh_elems[d]));
        }
    }

    {  // Free resources on GPU
        SFEM_CUDA_CHECK(hipFree(de_xyz));
        SFEM_CUDA_CHECK(hipFree(de_matrix));
        SFEM_CUDA_CHECK(hipFree(d_fff));

        for (int d = 0; d < 4; d++) {
            SFEM_CUDA_CHECK(hipFree(hd_elems[d]));
        }
        SFEM_CUDA_CHECK(hipFree(d_elems));

        crs_device_free(d_rowptr, d_colidx, d_values);

        for (int s = 0; s < nstreams; s++) {
            hipStreamDestroy(stream[s]);
            hipEventDestroy(event[s]);
        }
    }

    SFEM_RANGE_POP();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("cuda_laplacian_3.c: laplacian_assemble_hessian\t%g seconds\nloops %d\n",
           milliseconds / 1000,
           int(nelements / nbatch));
}
