#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
// #include <cstdio>
#include <algorithm>
#include <cstddef>

extern "C" {
#include "sfem_base.h"
#include "sfem_vec.h"
#include "sortreduce.h"
}
#include "sfem_cuda_base.h"
#include "sfem_defs.h"
#include "tet10_linear_elasticity_incore_cuda.h"

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define POW2(a) ((a) * (a))

// #define SFEM_ENABLE_FP32_KERNELS

#ifdef SFEM_ENABLE_FP32_KERNELS
typedef float scalar_t;
#else
typedef real_t scalar_t;
#endif

#ifdef SFEM_ENABLE_FP16_JACOBIANS
#include <hip/hip_fp16.h>
typedef half cu_jacobian_t;
#else
typedef geom_t cu_jacobian_t;
#endif

static inline __device__ __host__ void adjugate_and_det_micro_kernel(
    const geom_t px0,
    const geom_t px1,
    const geom_t px2,
    const geom_t px3,
    const geom_t py0,
    const geom_t py1,
    const geom_t py2,
    const geom_t py3,
    const geom_t pz0,
    const geom_t pz1,
    const geom_t pz2,
    const geom_t pz3,
    const ptrdiff_t stride,
    jacobian_t *adjugate,
    jacobian_t *jacobian_determinant) {
    // Compute jacobian in high precision
    real_t jacobian[9];
    jacobian[0] = -px0 + px1;
    jacobian[1] = -px0 + px2;
    jacobian[2] = -px0 + px3;
    jacobian[3] = -py0 + py1;
    jacobian[4] = -py0 + py2;
    jacobian[5] = -py0 + py3;
    jacobian[6] = -pz0 + pz1;
    jacobian[7] = -pz0 + pz2;
    jacobian[8] = -pz0 + pz3;

    const real_t x0 = jacobian[4] * jacobian[8];
    const real_t x1 = jacobian[5] * jacobian[7];
    const real_t x2 = jacobian[1] * jacobian[8];
    const real_t x3 = jacobian[1] * jacobian[5];
    const real_t x4 = jacobian[2] * jacobian[4];

    // Store adjugate in lower precision
    adjugate[0 * stride] = x0 - x1;
    adjugate[1 * stride] = jacobian[2] * jacobian[7] - x2;
    adjugate[2 * stride] = x3 - x4;
    adjugate[3 * stride] = -jacobian[3] * jacobian[8] + jacobian[5] * jacobian[6];
    adjugate[4 * stride] = jacobian[0] * jacobian[8] - jacobian[2] * jacobian[6];
    adjugate[5 * stride] = -jacobian[0] * jacobian[5] + jacobian[2] * jacobian[3];
    adjugate[6 * stride] = jacobian[3] * jacobian[7] - jacobian[4] * jacobian[6];
    adjugate[7 * stride] = -jacobian[0] * jacobian[7] + jacobian[1] * jacobian[6];
    adjugate[8 * stride] = jacobian[0] * jacobian[4] - jacobian[1] * jacobian[3];

    // Store determinant in lower precision
    jacobian_determinant[0] = jacobian[0] * x0 - jacobian[0] * x1 +
                              jacobian[2] * jacobian[3] * jacobian[7] - jacobian[3] * x2 +
                              jacobian[6] * x3 - jacobian[6] * x4;
}

static inline __device__ __host__ void ref_shape_grad_x(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qx;
    const scalar_t x1 = 4 * qy;
    const scalar_t x2 = 4 * qz;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = x0 - 1;
    out[2] = 0;
    out[3] = 0;
    out[4] = -8 * qx - x3 + 4;
    out[5] = x1;
    out[6] = -x1;
    out[7] = -x2;
    out[8] = x2;
    out[9] = 0;
}

static inline __device__ __host__ void ref_shape_grad_y(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qy;
    const scalar_t x1 = 4 * qx;
    const scalar_t x2 = 4 * qz;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = 0;
    out[2] = x0 - 1;
    out[3] = 0;
    out[4] = -x1;
    out[5] = x1;
    out[6] = -8 * qy - x3 + 4;
    out[7] = -x2;
    out[8] = 0;
    out[9] = x2;
}

static inline __device__ __host__ void ref_shape_grad_z(const scalar_t qx,
                                                        const scalar_t qy,
                                                        const scalar_t qz,
                                                        scalar_t *const out) {
    const scalar_t x0 = 4 * qz;
    const scalar_t x1 = 4 * qx;
    const scalar_t x2 = 4 * qy;
    const scalar_t x3 = x1 + x2;
    out[0] = x0 + x3 - 3;
    out[1] = 0;
    out[2] = 0;
    out[3] = x0 - 1;
    out[4] = -x1;
    out[5] = 0;
    out[6] = -x2;
    out[7] = -8 * qz - x3 + 4;
    out[8] = x1;
    out[9] = x2;
}

static inline __device__ __host__ void apply_micro_kernel(
    const scalar_t mu,
    const scalar_t lambda,
    const scalar_t *const SFEM_RESTRICT adjugate,
    const scalar_t jacobian_determinant,
    const scalar_t qx,
    const scalar_t qy,
    const scalar_t qz,
    const scalar_t qw,
    const scalar_t *const SFEM_RESTRICT u,
    scalar_t *const SFEM_RESTRICT element_vector) {
    // This can be reduced with 1D products (ref_shape_grad_{x,y,z})
    scalar_t disp_grad[9] = {0};

#define MICRO_KERNEL_USE_CODEGEN 1

#if MICRO_KERNEL_USE_CODEGEN
    // Code-gen way

    const scalar_t denom = 1;
    {
        const scalar_t x0 = 1.0 / jacobian_determinant;
        const scalar_t x1 = 4 * qx;
        const scalar_t x2 = x1 - 1;
        const scalar_t x3 = 4 * qy;
        const scalar_t x4 = -u[6] * x3;
        const scalar_t x5 = qz - 1;
        const scalar_t x6 = 8 * qx + 4 * qy + 4 * x5;
        const scalar_t x7 = 4 * qz;
        const scalar_t x8 = x1 + x3 + x7 - 3;
        const scalar_t x9 = u[0] * x8;
        const scalar_t x10 = -u[7] * x7 + x9;
        const scalar_t x11 = u[1] * x2 - u[4] * x6 + u[5] * x3 + u[8] * x7 + x10 + x4;
        const scalar_t x12 = x3 - 1;
        const scalar_t x13 = -u[4] * x1;
        const scalar_t x14 = 4 * qx + 8 * qy + 4 * x5;
        const scalar_t x15 = u[2] * x12 + u[5] * x1 - u[6] * x14 + u[9] * x7 + x10 + x13;
        const scalar_t x16 = x7 - 1;
        const scalar_t x17 = 4 * qx + 4 * qy + 8 * qz - 4;
        const scalar_t x18 = u[3] * x16 - u[7] * x17 + u[8] * x1 + u[9] * x3 + x13 + x4 + x9;
        const scalar_t x19 = -u[16] * x3;
        const scalar_t x20 = u[10] * x8;
        const scalar_t x21 = -u[17] * x7 + x20;
        const scalar_t x22 = u[11] * x2 - u[14] * x6 + u[15] * x3 + u[18] * x7 + x19 + x21;
        const scalar_t x23 = -u[14] * x1;
        const scalar_t x24 = u[12] * x12 + u[15] * x1 - u[16] * x14 + u[19] * x7 + x21 + x23;
        const scalar_t x25 = u[13] * x16 - u[17] * x17 + u[18] * x1 + u[19] * x3 + x19 + x20 + x23;
        const scalar_t x26 = -u[26] * x3;
        const scalar_t x27 = u[20] * x8;
        const scalar_t x28 = -u[27] * x7 + x27;
        const scalar_t x29 = u[21] * x2 - u[24] * x6 + u[25] * x3 + u[28] * x7 + x26 + x28;
        const scalar_t x30 = -u[24] * x1;
        const scalar_t x31 = u[22] * x12 + u[25] * x1 - u[26] * x14 + u[29] * x7 + x28 + x30;
        const scalar_t x32 = u[23] * x16 - u[27] * x17 + u[28] * x1 + u[29] * x3 + x26 + x27 + x30;
        disp_grad[0] = x0 * (adjugate[0] * x11 + adjugate[3] * x15 + adjugate[6] * x18);
        disp_grad[1] = x0 * (adjugate[1] * x11 + adjugate[4] * x15 + adjugate[7] * x18);
        disp_grad[2] = x0 * (adjugate[2] * x11 + adjugate[5] * x15 + adjugate[8] * x18);
        disp_grad[3] = x0 * (adjugate[0] * x22 + adjugate[3] * x24 + adjugate[6] * x25);
        disp_grad[4] = x0 * (adjugate[1] * x22 + adjugate[4] * x24 + adjugate[7] * x25);
        disp_grad[5] = x0 * (adjugate[2] * x22 + adjugate[5] * x24 + adjugate[8] * x25);
        disp_grad[6] = x0 * (adjugate[0] * x29 + adjugate[3] * x31 + adjugate[6] * x32);
        disp_grad[7] = x0 * (adjugate[1] * x29 + adjugate[4] * x31 + adjugate[7] * x32);
        disp_grad[8] = x0 * (adjugate[2] * x29 + adjugate[5] * x31 + adjugate[8] * x32);
    }
#else
    // Programmatic way

    const scalar_t denom = jacobian_determinant;
    {
        scalar_t temp[9] = {0};
        scalar_t grad[10];

        ref_shape_grad_x(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[0] += u[i] * g;
            temp[3] += u[10 + i] * g;
            temp[6] += u[20 + i] * g;
        }

        ref_shape_grad_y(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[1] += u[i] * g;
            temp[4] += u[10 + i] * g;
            temp[7] += u[20 + i] * g;
        }

        ref_shape_grad_z(qx, qy, qz, grad);
#pragma unroll
        for (int i = 0; i < 10; i++) {
            const scalar_t g = grad[i];
            temp[2] += u[i] * g;
            temp[5] += u[10 + i] * g;
            temp[8] += u[20 + i] * g;
        }

        for (int i = 0; i < 3; i++) {
#pragma unroll
            for (int j = 0; j < 3; j++) {
#pragma unroll
                for (int k = 0; k < 3; k++) {
                    disp_grad[i * 3 + j] += temp[i * 3 + k] * adjugate[k * 3 + j];
                }
            }
        }
    }

#endif
    // Includes first Piola-Kirchoff stress: P^T * J^-T * det(J)

    scalar_t *P_tXJinv_t = disp_grad;
    {
        const scalar_t x0 = (1.0 / 6.0) * mu;
        const scalar_t x1 = x0 * (disp_grad[1] + disp_grad[3]);
        const scalar_t x2 = x0 * (disp_grad[2] + disp_grad[6]);
        const scalar_t x3 = 2 * mu;
        const scalar_t x4 = lambda * (disp_grad[0] + disp_grad[4] + disp_grad[8]);
        const scalar_t x5 = (1.0 / 6.0) * disp_grad[0] * x3 + (1.0 / 6.0) * x4;
        const scalar_t x6 = x0 * (disp_grad[5] + disp_grad[7]);
        const scalar_t x7 = (1.0 / 6.0) * disp_grad[4] * x3 + (1.0 / 6.0) * x4;
        const scalar_t x8 = (1.0 / 6.0) * disp_grad[8] * x3 + (1.0 / 6.0) * x4;
        P_tXJinv_t[0] = adjugate[0] * x5 + adjugate[1] * x1 + adjugate[2] * x2;
        P_tXJinv_t[1] = adjugate[3] * x5 + adjugate[4] * x1 + adjugate[5] * x2;
        P_tXJinv_t[2] = adjugate[6] * x5 + adjugate[7] * x1 + adjugate[8] * x2;
        P_tXJinv_t[3] = adjugate[0] * x1 + adjugate[1] * x7 + adjugate[2] * x6;
        P_tXJinv_t[4] = adjugate[3] * x1 + adjugate[4] * x7 + adjugate[5] * x6;
        P_tXJinv_t[5] = adjugate[6] * x1 + adjugate[7] * x7 + adjugate[8] * x6;
        P_tXJinv_t[6] = adjugate[0] * x2 + adjugate[1] * x6 + adjugate[2] * x8;
        P_tXJinv_t[7] = adjugate[3] * x2 + adjugate[4] * x6 + adjugate[5] * x8;
        P_tXJinv_t[8] = adjugate[6] * x2 + adjugate[7] * x6 + adjugate[8] * x8;
    }

    // Scale by quadrature weight
    for (int i = 0; i < 9; i++) {
        P_tXJinv_t[i] *= qw / denom;
    }

// On CPU both versions are equivalent
#if MICRO_KERNEL_USE_CODEGEN
    {
        const scalar_t x0 = 4 * qx;
        const scalar_t x1 = 4 * qy;
        const scalar_t x2 = 4 * qz;
        const scalar_t x3 = x0 + x1 + x2 - 3;
        const scalar_t x4 = x0 - 1;
        const scalar_t x5 = x1 - 1;
        const scalar_t x6 = x2 - 1;
        const scalar_t x7 = P_tXJinv_t[1] * x0;
        const scalar_t x8 = P_tXJinv_t[2] * x0;
        const scalar_t x9 = qz - 1;
        const scalar_t x10 = 8 * qx + 4 * qy + 4 * x9;
        const scalar_t x11 = P_tXJinv_t[0] * x1;
        const scalar_t x12 = P_tXJinv_t[2] * x1;
        const scalar_t x13 = 4 * qx + 8 * qy + 4 * x9;
        const scalar_t x14 = P_tXJinv_t[0] * x2;
        const scalar_t x15 = P_tXJinv_t[1] * x2;
        const scalar_t x16 = 4 * qx + 4 * qy + 8 * qz - 4;
        const scalar_t x17 = P_tXJinv_t[4] * x0;
        const scalar_t x18 = P_tXJinv_t[5] * x0;
        const scalar_t x19 = P_tXJinv_t[3] * x1;
        const scalar_t x20 = P_tXJinv_t[5] * x1;
        const scalar_t x21 = P_tXJinv_t[3] * x2;
        const scalar_t x22 = P_tXJinv_t[4] * x2;
        const scalar_t x23 = P_tXJinv_t[7] * x0;
        const scalar_t x24 = P_tXJinv_t[8] * x0;
        const scalar_t x25 = P_tXJinv_t[6] * x1;
        const scalar_t x26 = P_tXJinv_t[8] * x1;
        const scalar_t x27 = P_tXJinv_t[6] * x2;
        const scalar_t x28 = P_tXJinv_t[7] * x2;
        element_vector[0] += x3 * (P_tXJinv_t[0] + P_tXJinv_t[1] + P_tXJinv_t[2]);
        element_vector[1] += P_tXJinv_t[0] * x4;
        element_vector[2] += P_tXJinv_t[1] * x5;
        element_vector[3] += P_tXJinv_t[2] * x6;
        element_vector[4] += -P_tXJinv_t[0] * x10 - x7 - x8;
        element_vector[5] += x11 + x7;
        element_vector[6] += -P_tXJinv_t[1] * x13 - x11 - x12;
        element_vector[7] += -P_tXJinv_t[2] * x16 - x14 - x15;
        element_vector[8] += x14 + x8;
        element_vector[9] += x12 + x15;
        element_vector[10] += x3 * (P_tXJinv_t[3] + P_tXJinv_t[4] + P_tXJinv_t[5]);
        element_vector[11] += P_tXJinv_t[3] * x4;
        element_vector[12] += P_tXJinv_t[4] * x5;
        element_vector[13] += P_tXJinv_t[5] * x6;
        element_vector[14] += -P_tXJinv_t[3] * x10 - x17 - x18;
        element_vector[15] += x17 + x19;
        element_vector[16] += -P_tXJinv_t[4] * x13 - x19 - x20;
        element_vector[17] += -P_tXJinv_t[5] * x16 - x21 - x22;
        element_vector[18] += x18 + x21;
        element_vector[19] += x20 + x22;
        element_vector[20] += x3 * (P_tXJinv_t[6] + P_tXJinv_t[7] + P_tXJinv_t[8]);
        element_vector[21] += P_tXJinv_t[6] * x4;
        element_vector[22] += P_tXJinv_t[7] * x5;
        element_vector[23] += P_tXJinv_t[8] * x6;
        element_vector[24] += -P_tXJinv_t[6] * x10 - x23 - x24;
        element_vector[25] += x23 + x25;
        element_vector[26] += -P_tXJinv_t[7] * x13 - x25 - x26;
        element_vector[27] += -P_tXJinv_t[8] * x16 - x27 - x28;
        element_vector[28] += x24 + x27;
        element_vector[29] += x26 + x28;
    }

#else

    {
        scalar_t grad[10];
        ref_shape_grad_x(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += P_tXJinv_t[0] * g;
            element_vector[10 + i] += P_tXJinv_t[3] * g;
            element_vector[20 + i] += P_tXJinv_t[6] * g;
        }

        ref_shape_grad_y(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += P_tXJinv_t[1] * g;
            element_vector[10 + i] += P_tXJinv_t[4] * g;
            element_vector[20 + i] += P_tXJinv_t[7] * g;
        }

        ref_shape_grad_z(qx, qy, qz, grad);

#pragma unroll
        for (int i = 0; i < 10; i++) {
            scalar_t g = grad[i];
            element_vector[i] += P_tXJinv_t[2] * g;
            element_vector[10 + i] += P_tXJinv_t[5] * g;
            element_vector[20 + i] += P_tXJinv_t[8] * g;
        }
    }

#endif

#undef MICRO_KERNEL_USE_CODEGEN
}

static const int n_qp = 8;
static const scalar_t h_qx[8] =
    {0.0, 1.0, 0.0, 0.0, 0.333333333333, 0.333333333333, 0.0, 0.333333333333};

static const scalar_t h_qy[8] =
    {0.0, 0.0, 1.0, 0.0, 0.333333333333, 0.0, 0.333333333333, 0.333333333333};

static const scalar_t h_qz[8] =
    {0.0, 0.0, 0.0, 1.0, 0.0, 0.333333333333, 0.333333333333, 0.333333333333};

static const scalar_t h_qw[8] = {0.025, 0.025, 0.025, 0.025, 0.225, 0.225, 0.225, 0.225};

__constant__ scalar_t qx[8];
__constant__ scalar_t qy[8];
__constant__ scalar_t qz[8];
__constant__ scalar_t qw[8];

static void init_quadrature() {
    static bool initialized = false;
    if (initialized) {
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qx), h_qx, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qy), h_qy, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qz), h_qz, 8 * sizeof(scalar_t)));
        SFEM_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(qw), h_qw, 8 * sizeof(scalar_t)));
    }
}

int tet10_cuda_incore_linear_elasticity_init(cuda_incore_linear_elasticity_t *const ctx,
                                             const real_t mu,
                                             const real_t lambda,
                                             const ptrdiff_t nelements,
                                             idx_t **const SFEM_RESTRICT elements,
                                             geom_t **const SFEM_RESTRICT points) {
    {
        init_quadrature();
        cu_jacobian_t *jacobian_adjugate =
            (cu_jacobian_t *)calloc(9 * nelements, sizeof(cu_jacobian_t));
        cu_jacobian_t *jacobian_determinant =
            (cu_jacobian_t *)calloc(nelements, sizeof(cu_jacobian_t));

#pragma omp parallel
        {
#pragma omp for
            for (ptrdiff_t e = 0; e < nelements; e++) {
                adjugate_and_det_micro_kernel(points[0][elements[0][e]],
                                              points[0][elements[1][e]],
                                              points[0][elements[2][e]],
                                              points[0][elements[3][e]],
                                              points[1][elements[0][e]],
                                              points[1][elements[1][e]],
                                              points[1][elements[2][e]],
                                              points[1][elements[3][e]],
                                              points[2][elements[0][e]],
                                              points[2][elements[1][e]],
                                              points[2][elements[2][e]],
                                              points[2][elements[3][e]],
                                              nelements,
                                              &jacobian_adjugate[e],
                                              &jacobian_determinant[e]);
            }
        }

        SFEM_CUDA_CHECK(hipMalloc(&ctx->jacobian_adjugate, 9 * nelements * sizeof(cu_jacobian_t)));
        SFEM_CUDA_CHECK(hipMemcpy(ctx->jacobian_adjugate,
                                   jacobian_adjugate,
                                   9 * nelements * sizeof(cu_jacobian_t),
                                   hipMemcpyHostToDevice));
        free(jacobian_adjugate);

        SFEM_CUDA_CHECK(hipMalloc(&ctx->jacobian_determinant, nelements * sizeof(cu_jacobian_t)));
        SFEM_CUDA_CHECK(hipMemcpy(ctx->jacobian_determinant,
                                   jacobian_determinant,
                                   nelements * sizeof(cu_jacobian_t),
                                   hipMemcpyHostToDevice));
        free(jacobian_determinant);
    }

    {
        // Store elem indices on device
        SFEM_CUDA_CHECK(hipMalloc(&ctx->elements, 10 * nelements * sizeof(idx_t)));

        for (int d = 0; d < 10; d++) {
            SFEM_CUDA_CHECK(hipMemcpy(ctx->elements + d * nelements,
                                       elements[d],
                                       nelements * sizeof(idx_t),
                                       hipMemcpyHostToDevice));
        }
    }

    ctx->mu = mu;
    ctx->lambda = lambda;
    ctx->nelements = nelements;
    ctx->element_type = TET4;

    return 0;
}

int tet10_cuda_incore_linear_elasticity_destroy(cuda_incore_linear_elasticity_t *const ctx) {
    hipFree(ctx->jacobian_adjugate);
    hipFree(ctx->jacobian_determinant);

    ctx->jacobian_adjugate = 0;
    ctx->jacobian_determinant = 0;

    ctx->elements = 0;
    ctx->nelements = 0;
    ctx->element_type = INVALID;
    return 0;
}

__global__ void tet10_cuda_incore_linear_elasticity_apply_opt_kernel(
    const ptrdiff_t nelements,
    idx_t *const elements,
    const cu_jacobian_t *const g_jacobian_adjugate,
    const cu_jacobian_t *const g_jacobian_determinant,
    const scalar_t mu,
    const scalar_t lambda,
    const real_t *const u,
    real_t *const values) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        idx_t ev[10];

        // Sub-geometry
        scalar_t adjugate[9];
        scalar_t element_u[30];
        scalar_t element_vector[30] = {0};
        ;

        // Copy over jacobian adjugate
        {
            const cu_jacobian_t *const jacobian_adjugate = &g_jacobian_adjugate[e];
            for (int i = 0; i < 9; i++) {
                adjugate[i] = jacobian_adjugate[i * nelements];
            }
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        const scalar_t jacobian_determinant = 1;
#else
        const scalar_t jacobian_determinant = g_jacobian_determinant[e];
#endif

#pragma unroll(10)
        for (int v = 0; v < 10; ++v) {
            ev[v] = elements[v * nelements + e];
        }

        for (int v = 0; v < 10; ++v) {
            element_u[v] = u[ev[v] * 3];
            element_u[10 + v] = u[ev[v] * 3 + 1];
            element_u[20 + v] = u[ev[v] * 3 + 2];
        }

        for (int k = 0; k < n_qp; k++) {
            apply_micro_kernel(mu,
                               lambda,
                               adjugate,
                               jacobian_determinant,
                               qx[k],
                               qy[k],
                               qz[k],
                               qw[k],
                               element_u,
                               element_vector);
        }

#ifdef SFEM_ENABLE_FP32_KERNELS
        //
        {
            // real_t use here instead of scalar_t to have division in full precision
            const real_t jacobian_determinant = g_jacobian_determinant[e];

            for (int v = 0; v < 10; v++) {
                atomicAdd(&values[ev[v] * 3], element_vector[v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&values[ev[v] * 3 + 1], element_vector[10 + v] / jacobian_determinant);
            }

            for (int v = 0; v < 10; v++) {
                atomicAdd(&values[ev[v] * 3 + 2], element_vector[20 + v] / jacobian_determinant);
            }
        }
#else

        for (int v = 0; v < 10; v++) {
            atomicAdd(&values[ev[v] * 3], element_vector[v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&values[ev[v] * 3 + 1], element_vector[10 + v]);
        }

        for (int v = 0; v < 10; v++) {
            atomicAdd(&values[ev[v] * 3 + 2], element_vector[20 + v]);
        }
#endif
    }
}

#define SFEM_USE_OCCUPANCY_MAX_POTENTIAL

int tet10_cuda_incore_linear_elasticity_apply_opt(const cuda_incore_linear_elasticity_t *const ctx,
                                                  const real_t *const SFEM_RESTRICT u,
                                                  real_t *const SFEM_RESTRICT values) {
    const real_t mu = ctx->mu;
    const real_t lambda = ctx->lambda;

    const cu_jacobian_t *const jacobian_adjugate = (cu_jacobian_t *)ctx->jacobian_adjugate;
    const cu_jacobian_t *const jacobian_determinant = (cu_jacobian_t *)ctx->jacobian_determinant;

    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                           &block_size,
                                           tet10_cuda_incore_linear_elasticity_apply_opt_kernel,
                                           0,
                                           0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = std::max(ptrdiff_t(1), (ctx->nelements + block_size - 1) / block_size);
    tet10_cuda_incore_linear_elasticity_apply_opt_kernel<<<n_blocks, block_size, 0>>>(
        ctx->nelements,
        ctx->elements,
        jacobian_adjugate,
        jacobian_determinant,
        mu,
        lambda,
        u,
        values);

    return 0;
}

int tet10_cuda_incore_linear_elasticity_diag(const cuda_incore_linear_elasticity_t *const ctx,
                                             real_t *const SFEM_RESTRICT diag) {
    //
    assert(0);
    return 1;
}

int tet10_cuda_incore_linear_elasticity_apply_aos(const ptrdiff_t nelements,
                                                  const ptrdiff_t nnodes,
                                                  idx_t **const SFEM_RESTRICT elements,
                                                  geom_t **const SFEM_RESTRICT points,
                                                  const real_t mu,
                                                  const real_t lambda,
                                                  const real_t *const SFEM_RESTRICT u,
                                                  real_t *const SFEM_RESTRICT values) {
    cuda_incore_linear_elasticity_t ctx;
    tet10_cuda_incore_linear_elasticity_init(&ctx, mu, lambda, nelements, elements, points);
    tet10_cuda_incore_linear_elasticity_apply_opt(&ctx, u, values);
    tet10_cuda_incore_linear_elasticity_destroy(&ctx);
    return 0;
}
