#include "hip/hip_runtime.h"
#include "sfem_base.h"

#ifdef __NVCC__
#include "sfem_cuda_base.h"
#define SFEM_DEVICE_FUNCTION static SFEM_INLINE __device__ __host__
#else
#define SFEM_DEVICE_FUNCTION static SFEM_INLINE
#endif 

static const int fe_spatial_dim = 3;
static const int fe_manifold_dim = 2;
static const int fe_n_nodes = 6;
static const char * fe_name = "TriShell6";
static const int fe_n_nodes_for_jacobian = 3;
static const int fe_subparam_n_nodes = 3;


SFEM_DEVICE_FUNCTION void TriShell6_mk_jacobian(
const real_t px0,
const real_t px1,
const real_t px2,
const real_t py0,
const real_t py1,
const real_t py2,
const real_t pz0,
const real_t pz1,
const real_t pz2,
const count_t stride_jacobian,
real_t *jacobian
)
{
jacobian[0*stride_jacobian] = -px0 + px1;
jacobian[1*stride_jacobian] = -px0 + px2;
jacobian[2*stride_jacobian] = -py0 + py1;
jacobian[3*stride_jacobian] = -py0 + py2;
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_jacobian_inverse(
const real_t px0,
const real_t px1,
const real_t px2,
const real_t py0,
const real_t py1,
const real_t py2,
const real_t pz0,
const real_t pz1,
const real_t pz2,
const count_t stride_jacobian_inverse,
real_t *jacobian_inverse
)
{
const real_t x0 = -px0 + px1;
const real_t x1 = -px0 + px2;
const real_t x2 = -py0 + py2;
const real_t x3 = -pz0 + pz2;
const real_t x4 = pow(x1, 2) + pow(x2, 2) + pow(x3, 2);
const real_t x5 = -py0 + py1;
const real_t x6 = -pz0 + pz1;
const real_t x7 = x0*x1 + x2*x5 + x3*x6;
const real_t x8 = pow(x0, 2) + pow(x5, 2) + pow(x6, 2);
const real_t x9 = 1.0/(x4*x8 - pow(x7, 2));
const real_t x10 = x4*x9;
const real_t x11 = -x7*x9;
const real_t x12 = x8*x9;
jacobian_inverse[0*stride_jacobian_inverse] = x0*x10 + x1*x11;
jacobian_inverse[1*stride_jacobian_inverse] = x10*x5 + x11*x2;
jacobian_inverse[2*stride_jacobian_inverse] = x10*x6 + x11*x3;
jacobian_inverse[3*stride_jacobian_inverse] = x0*x11 + x1*x12;
jacobian_inverse[4*stride_jacobian_inverse] = x11*x5 + x12*x2;
jacobian_inverse[5*stride_jacobian_inverse] = x11*x6 + x12*x3;
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_jacobian_determinant(
const real_t px0,
const real_t px1,
const real_t px2,
const real_t py0,
const real_t py1,
const real_t py2,
const real_t pz0,
const real_t pz1,
const real_t pz2,
 // arrays
const count_t stride_jacobian_determinant,
real_t *jacobian_determinant
)
{
const real_t x0 = -px0 + px1;
const real_t x1 = -px0 + px2;
const real_t x2 = -py0 + py1;
const real_t x3 = -py0 + py2;
const real_t x4 = -pz0 + pz1;
const real_t x5 = -pz0 + pz2;
jacobian_determinant[0] = sqrt((pow(x0, 2) + pow(x2, 2) + pow(x4, 2))*(pow(x1, 2) + pow(x3, 2) + pow(x5, 2)) - pow(x0*x1 + x2*x3 + x4*x5, 2));
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_fun(
const real_t qx,
const real_t qy,
 // arrays
const int stride_fun,
real_t * SFEM_RESTRICT f
)
{
const real_t x0 = 4*qx*qy;
const real_t x1 = pow(qx, 2);
const real_t x2 = 2*x1;
const real_t x3 = pow(qy, 2);
const real_t x4 = 2*x3;
f[0*stride_fun] = -3*qx - 3*qy + x0 + x2 + x4 + 1;
f[1*stride_fun] = -qx + x2;
f[2*stride_fun] = -qy + x4;
f[3*stride_fun] = 4*qx - x0 - 4*x1;
f[4*stride_fun] = x0;
f[5*stride_fun] = 4*qy - x0 - 4*x3;
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_partial_x(
const real_t qx,
const real_t qy,
 // arrays
const count_t stride_jacobian_inverse,
const real_t * SFEM_RESTRICT jacobian_inverse,
const count_t stride_grad,
real_t * SFEM_RESTRICT gx
)
{
const real_t x0 = 4*qx;
const real_t x1 = 4*qy;
const real_t x2 = x0 + x1 - 3;
const real_t x3 = jacobian_inverse[3*stride_jacobian_inverse]*x0;
const real_t x4 = jacobian_inverse[0*stride_jacobian_inverse]*x1;
gx[0*stride_grad] = jacobian_inverse[0*stride_jacobian_inverse]*x2 + jacobian_inverse[3*stride_jacobian_inverse]*x2;
gx[1*stride_grad] = jacobian_inverse[0*stride_jacobian_inverse]*(x0 - 1);
gx[2*stride_grad] = jacobian_inverse[3*stride_jacobian_inverse]*(x1 - 1);
gx[3*stride_grad] = jacobian_inverse[0*stride_jacobian_inverse]*(-8*qx - x1 + 4) - x3;
gx[4*stride_grad] = x3 + x4;
gx[5*stride_grad] = jacobian_inverse[3*stride_jacobian_inverse]*(-8*qy - x0 + 4) - x4;
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_partial_y(
const real_t qx,
const real_t qy,
 // arrays
const count_t stride_jacobian_inverse,
const real_t * SFEM_RESTRICT jacobian_inverse,
const count_t stride_grad,
real_t * SFEM_RESTRICT gy
)
{
const real_t x0 = 4*qx;
const real_t x1 = 4*qy;
const real_t x2 = x0 + x1 - 3;
const real_t x3 = jacobian_inverse[4*stride_jacobian_inverse]*x0;
const real_t x4 = jacobian_inverse[1*stride_jacobian_inverse]*x1;
gy[0*stride_grad] = jacobian_inverse[1*stride_jacobian_inverse]*x2 + jacobian_inverse[4*stride_jacobian_inverse]*x2;
gy[1*stride_grad] = jacobian_inverse[1*stride_jacobian_inverse]*(x0 - 1);
gy[2*stride_grad] = jacobian_inverse[4*stride_jacobian_inverse]*(x1 - 1);
gy[3*stride_grad] = jacobian_inverse[1*stride_jacobian_inverse]*(-8*qx - x1 + 4) - x3;
gy[4*stride_grad] = x3 + x4;
gy[5*stride_grad] = jacobian_inverse[4*stride_jacobian_inverse]*(-8*qy - x0 + 4) - x4;
}

SFEM_DEVICE_FUNCTION void TriShell6_mk_partial_z(
const real_t qx,
const real_t qy,
 // arrays
const count_t stride_jacobian_inverse,
const real_t * SFEM_RESTRICT jacobian_inverse,
const count_t stride_grad,
real_t * SFEM_RESTRICT gz
) 
{
const real_t x0 = 4*qx;
const real_t x1 = 4*qy;
const real_t x2 = x0 + x1 - 3;
const real_t x3 = jacobian_inverse[5*stride_jacobian_inverse]*x0;
const real_t x4 = jacobian_inverse[2*stride_jacobian_inverse]*x1;
gz[0*stride_grad] = jacobian_inverse[2*stride_jacobian_inverse]*x2 + jacobian_inverse[5*stride_jacobian_inverse]*x2;
gz[1*stride_grad] = jacobian_inverse[2*stride_jacobian_inverse]*(x0 - 1);
gz[2*stride_grad] = jacobian_inverse[5*stride_jacobian_inverse]*(x1 - 1);
gz[3*stride_grad] = jacobian_inverse[2*stride_jacobian_inverse]*(-8*qx - x1 + 4) - x3;
gz[4*stride_grad] = x3 + x4;
gz[5*stride_grad] = jacobian_inverse[5*stride_jacobian_inverse]*(-8*qy - x0 + 4) - x4;
}
