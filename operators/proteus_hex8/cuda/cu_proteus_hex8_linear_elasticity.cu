#include "hip/hip_runtime.h"
#include "cu_hex8_linear_elasticity_inline.hpp"
#include "cu_proteus_hex8_linear_elasticity.h"
#include "sfem_cuda_base.h"

#ifndef MAX
#define MAX(a, b) ((a) >= (b) ? (a) : (b))
#endif

#ifndef MIN
#define MIN(a, b) ((a) <= (b) ? (a) : (b))
#endif

#define B_(x, y, z) ((z)*BLOCK_SIZE_2 + (y)*BLOCK_SIZE + (x))

template <typename T, int LEVEL>
__global__ void cu_proteus_affine_hex8_linear_elasticity_apply_kernel(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT g_jacobian_determinant,
        const T mu,
        const T lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT g_ux,
        const T *const SFEM_RESTRICT g_uy,
        const T *const SFEM_RESTRICT g_uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT g_outx,
        T *const SFEM_RESTRICT g_outy,
        T *const SFEM_RESTRICT g_outz) {
    static const int BLOCK_SIZE = LEVEL + 1;
    static const int BLOCK_SIZE_2 = BLOCK_SIZE * BLOCK_SIZE;
    static const int BLOCK_SIZE_3 = BLOCK_SIZE_2 * BLOCK_SIZE;

    assert(blockDim.x == BLOCK_SIZE);
    assert(blockDim.y == BLOCK_SIZE);
    assert(blockDim.z == BLOCK_SIZE);

    __shared__ T ux_block[BLOCK_SIZE_3];
    __shared__ T uy_block[BLOCK_SIZE_3];
    __shared__ T uz_block[BLOCK_SIZE_3];

    __shared__ T outx_block[BLOCK_SIZE_3];
    __shared__ T outy_block[BLOCK_SIZE_3];
    __shared__ T outz_block[BLOCK_SIZE_3];

    static const int n_qp = 6;
    const T qw[6] = {0.16666666666666666666666666666667,
                     0.16666666666666666666666666666667,
                     0.16666666666666666666666666666667,
                     0.16666666666666666666666666666667,
                     0.16666666666666666666666666666667,
                     0.16666666666666666666666666666667};
    const T qx[6] = {0.0, 0.5, 0.5, 0.5, 0.5, 1.0};
    const T qy[6] = {0.5, 0.0, 0.5, 0.5, 1.0, 0.5};
    const T qz[6] = {0.5, 0.5, 0.0, 1.0, 0.5, 0.5};

    for (ptrdiff_t e = blockIdx.x; e < nelements; e += gridDim.x) {
        const int lidx = threadIdx.z * BLOCK_SIZE_2 + threadIdx.y * BLOCK_SIZE + threadIdx.x;
        const ptrdiff_t idx = elements[lidx * stride + e];

        ux_block[lidx] = g_ux[idx * u_stride];
        uy_block[lidx] = g_uy[idx * u_stride];
        uz_block[lidx] = g_uz[idx * u_stride];

        outx_block[lidx] = 0;
        outy_block[lidx] = 0;
        outz_block[lidx] = 0;

        T sub_adjugate[9];
        T sub_determinant;

        {
            const T h = 1. / LEVEL;
            cu_hex8_sub_adj_0(stride,
                              &g_jacobian_adjugate[e],
                              g_jacobian_determinant[e],
                              h,
                              sub_adjugate,
                              &sub_determinant);
        }

        T outx[8] = {0};
        T outy[8] = {0};
        T outz[8] = {0};

        const bool is_element = threadIdx.x < LEVEL && threadIdx.y < LEVEL && threadIdx.z < LEVEL;

        __syncthreads();
        
        if (is_element) {
            const T ux[8] = {ux_block[B_(threadIdx.x, threadIdx.y, threadIdx.z)],
                             ux_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z)],
                             ux_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z)],
                             ux_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z)],
                             ux_block[B_(threadIdx.x, threadIdx.y, threadIdx.z + 1)],
                             ux_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z + 1)],
                             ux_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z + 1)],
                             ux_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z + 1)]};

            const T uy[8] = {uy_block[B_(threadIdx.x, threadIdx.y, threadIdx.z)],
                             uy_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z)],
                             uy_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z)],
                             uy_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z)],
                             uy_block[B_(threadIdx.x, threadIdx.y, threadIdx.z + 1)],
                             uy_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z + 1)],
                             uy_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z + 1)],
                             uy_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z + 1)]};

            const T uz[8] = {uz_block[B_(threadIdx.x, threadIdx.y, threadIdx.z)],
                             uz_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z)],
                             uz_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z)],
                             uz_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z)],
                             uz_block[B_(threadIdx.x, threadIdx.y, threadIdx.z + 1)],
                             uz_block[B_(threadIdx.x + 1, threadIdx.y, threadIdx.z + 1)],
                             uz_block[B_(threadIdx.x + 1, threadIdx.y + 1, threadIdx.z + 1)],
                             uz_block[B_(threadIdx.x, threadIdx.y + 1, threadIdx.z + 1)]};

            for (int k = 0; k < n_qp; k++) {
                cu_hex8_linear_elasticity_apply_adj(mu,
                                                    lambda,
                                                    sub_adjugate,
                                                    sub_determinant,
                                                    qx[k],
                                                    qy[k],
                                                    qz[k],
                                                    qw[k],
                                                    ux,
                                                    uy,
                                                    uz,
                                                    outx,
                                                    outy,
                                                    outz);
            }
        }

        const int interior = threadIdx.x > 0 && threadIdx.y > 0 && threadIdx.z > 0 &&
                             threadIdx.x < LEVEL && threadIdx.y < LEVEL && threadIdx.z < LEVEL;

        __syncthreads();

        if (interior) {
            g_outx[idx * out_stride] += outx_block[lidx];
            g_outy[idx * out_stride] += outy_block[lidx];
            g_outz[idx * out_stride] += outz_block[lidx];
        } else {
            atomicAdd(&g_outx[idx * out_stride], outx_block[lidx]);
            atomicAdd(&g_outy[idx * out_stride], outy_block[lidx]);
            atomicAdd(&g_outz[idx * out_stride], outz_block[lidx]);
        }
    }
}

template <typename T, int LEVEL>
int cu_proteus_affine_hex8_linear_elasticity_apply_warp_tpl(
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const T mu,
        const T lambda,
        const ptrdiff_t u_stride,
        const T *const SFEM_RESTRICT ux,
        const T *const SFEM_RESTRICT uy,
        const T *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        T *const SFEM_RESTRICT outx,
        T *const SFEM_RESTRICT outy,
        T *const SFEM_RESTRICT outz,
        void *stream) {
    SFEM_DEBUG_SYNCHRONIZE();

    static const int BLOCK_SIZE = LEVEL + 1;

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 n_blocks(MIN(nelements, 65535), 1, 1);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_proteus_affine_hex8_linear_elasticity_apply_kernel<T, LEVEL>
                <<<n_blocks, block_size, 0, s>>>(nelements,
                                                 stride,
                                                 interior_start,
                                                 elements,
                                                 jacobian_adjugate,
                                                 jacobian_determinant,
                                                 mu,
                                                 lambda,
                                                 u_stride,
                                                 ux,
                                                 uy,
                                                 uz,
                                                 out_stride,
                                                 outx,
                                                 outy,
                                                 outz);
    } else {
        cu_proteus_affine_hex8_linear_elasticity_apply_kernel<T, LEVEL>
                <<<n_blocks, block_size, 0>>>(nelements,
                                              stride,
                                              interior_start,
                                              elements,
                                              jacobian_adjugate,
                                              jacobian_determinant,
                                              mu,
                                              lambda,
                                              u_stride,
                                              ux,
                                              uy,
                                              uz,
                                              out_stride,
                                              outx,
                                              outy,
                                              outz);
    }

    SFEM_DEBUG_SYNCHRONIZE();
    return SFEM_SUCCESS;
}

template <typename real_t>
int cu_proteus_affine_hex8_linear_elasticity_apply_tpl(
        const int level,
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_adjugate,
        const cu_jacobian_t *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const ptrdiff_t u_stride,
        const real_t *const SFEM_RESTRICT ux,
        const real_t *const SFEM_RESTRICT uy,
        const real_t *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        real_t *const SFEM_RESTRICT outx,
        real_t *const SFEM_RESTRICT outy,
        real_t *const SFEM_RESTRICT outz,
        void *stream) {
    switch (level) {
        case 7: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_warp_tpl<real_t, 7>(
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (real_t *)ux,
                    (real_t *)uy,
                    (real_t *)uz,
                    out_stride,
                    (real_t *)outx,
                    (real_t *)outy,
                    (real_t *)outz,
                    stream);
        }
        case 8: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_warp_tpl<real_t, 8>(
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (real_t *)ux,
                    (real_t *)uy,
                    (real_t *)uz,
                    out_stride,
                    (real_t *)outx,
                    (real_t *)outy,
                    (real_t *)outz,
                    stream);
        }
        case 9: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_warp_tpl<real_t, 9>(
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (real_t *)ux,
                    (real_t *)uy,
                    (real_t *)uz,
                    out_stride,
                    (real_t *)outx,
                    (real_t *)outy,
                    (real_t *)outz,
                    stream);
        }
        case 10: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_warp_tpl<real_t, 10>(
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (real_t *)ux,
                    (real_t *)uy,
                    (real_t *)uz,
                    out_stride,
                    (real_t *)outx,
                    (real_t *)outy,
                    (real_t *)outz,
                    stream);
        }
        default: {
            fprintf(stderr,
                    "cu_proteus_affine_hex8_linear_elasticity_apply_tpl: level %d not supported!\n",
                    level);
            assert(false);
            return SFEM_FAILURE;
        }
    }
}

extern int cu_proteus_affine_hex8_linear_elasticity_apply(
        const int level,
        const ptrdiff_t nelements,
        const ptrdiff_t stride,  // Stride for elements and fff
        const ptrdiff_t interior_start,
        const idx_t *const SFEM_RESTRICT elements,
        const void *const SFEM_RESTRICT jacobian_adjugate,
        const void *const SFEM_RESTRICT jacobian_determinant,
        const real_t mu,
        const real_t lambda,
        const enum RealType real_type,
        const ptrdiff_t u_stride,
        const void *const SFEM_RESTRICT ux,
        const void *const SFEM_RESTRICT uy,
        const void *const SFEM_RESTRICT uz,
        const ptrdiff_t out_stride,
        void *const SFEM_RESTRICT outx,
        void *const SFEM_RESTRICT outy,
        void *const SFEM_RESTRICT outz,
        void *stream) {
    switch (real_type) {
        case SFEM_REAL_DEFAULT: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_tpl<real_t>(
                    level,
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (real_t *)ux,
                    (real_t *)uy,
                    (real_t *)uz,
                    out_stride,
                    (real_t *)outx,
                    (real_t *)outy,
                    (real_t *)outz,
                    stream);
        }
        case SFEM_FLOAT32: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_tpl<float>(
                    level,
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (float *)ux,
                    (float *)uy,
                    (float *)uz,
                    out_stride,
                    (float *)outx,
                    (float *)outy,
                    (float *)outz,
                    stream);
        }
        case SFEM_FLOAT64: {
            return cu_proteus_affine_hex8_linear_elasticity_apply_tpl<double>(
                    level,
                    nelements,
                    stride,
                    interior_start,
                    elements,
                    (cu_jacobian_t *)jacobian_adjugate,
                    (cu_jacobian_t *)jacobian_determinant,
                    mu,
                    lambda,
                    u_stride,
                    (double *)ux,
                    (double *)uy,
                    (double *)uz,
                    out_stride,
                    (double *)outx,
                    (double *)outy,
                    (double *)outz,
                    stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_proteus_affine_hex8_linear_elasticity_apply: not implemented for "
                    "type %s "
                    "(code %d)\n",
                    real_type_to_string(real_type),
                    real_type);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}
