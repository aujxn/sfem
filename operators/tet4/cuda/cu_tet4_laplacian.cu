#include "hip/hip_runtime.h"
#include "cu_tet4_laplacian.h"

#include "sfem_cuda_base.h"
#include "sfem_defs.h"

#include "cu_tet4_inline.hpp"
#include "cu_tet4_laplacian_inline.hpp"

#include <cassert>

template <typename real_t>
__global__ void cu_tet4_laplacian_apply_kernel(const ptrdiff_t nelements,
                                               const idx_t *const SFEM_RESTRICT elements,
                                               const cu_jacobian_t *const SFEM_RESTRICT fff,
                                               const real_t *const SFEM_RESTRICT x,
                                               real_t *const SFEM_RESTRICT y) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        scalar_t ex[4];
        scalar_t ey[4];
        idx_t vidx[4];

        // collect coeffs
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            vidx[v] = elements[v * nelements + e];
            ex[v] = x[vidx[v]];
        }

        scalar_t fffe[6];
#pragma unroll(6)
        for (int d = 0; d < 6; d++) {
            fffe[d] = fff[d * nelements + e];
        }

        tet4_laplacian_apply_fff(fffe, 1, ex, ey);

        // redistribute coeffs
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            atomicAdd(&y[vidx[v]], ey[v]);
        }
    }
}

template <typename T>
static int cu_tet4_laplacian_apply_tpl(const ptrdiff_t nelements,
                                       const idx_t *const SFEM_RESTRICT elements,
                                       const cu_jacobian_t *const SFEM_RESTRICT fff,
                                       const T *const x,
                                       T *const y,
                                       void *stream) {
    // Hand tuned
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(&min_grid_size,
                                           &block_size,
                                           cu_tet4_laplacian_apply_kernel<T>,
                                           0,
                                           0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    const ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_tet4_laplacian_apply_kernel<<<n_blocks, block_size, 0, s>>>(
                nelements, elements, fff, x, y);
    } else {
        cu_tet4_laplacian_apply_kernel<<<n_blocks, block_size, 0>>>(nelements, elements, fff, x, y);
    }

    return SFEM_SUCCESS;
}

extern int cu_tet4_laplacian_apply(const ptrdiff_t nelements,
                                   const idx_t *const SFEM_RESTRICT elements,
                                   const void *const SFEM_RESTRICT fff,
                                   const enum RealType real_type_xy,
                                   const void *const x,
                                   void *const y,
                                   void *stream) {
    switch (real_type_xy) {
        case SFEM_REAL_DEFAULT: {
            return cu_tet4_laplacian_apply_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (real_t *)x, (real_t *)y, stream);
        }
        case SFEM_FLOAT32: {
            return cu_tet4_laplacian_apply_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (float *)x, (float *)y, stream);
        }
        case SFEM_FLOAT64: {
            return cu_tet4_laplacian_apply_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (double *)x, (double *)y, stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet4_laplacian_apply: not implemented for type %s (code %d)\n",
                    real_type_to_string(real_type_xy),
                    real_type_xy);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}

template <typename real_t>
__global__ void cu_tet4_laplacian_diag_kernel(const ptrdiff_t nelements,
                                              const idx_t *const SFEM_RESTRICT elements,
                                              const cu_jacobian_t *const SFEM_RESTRICT fff,
                                              real_t *const SFEM_RESTRICT diag) {
    for (ptrdiff_t e = blockIdx.x * blockDim.x + threadIdx.x; e < nelements;
         e += blockDim.x * gridDim.x) {
        scalar_t ed[4];
        idx_t vidx[4];

        // collect coeffs
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            vidx[v] = elements[v * nelements + e];
        }

        scalar_t fffe[6];
#pragma unroll(6)
        for (int d = 0; d < 6; d++) {
            fffe[d] = fff[d * nelements + e];
        }

        // Assembler operator diagonal
        tet4_laplacian_diag_fff(fffe, 1, ed);

        // redistribute coeffs
#pragma unroll(4)
        for (int v = 0; v < 4; ++v) {
            atomicAdd(&diag[vidx[v]], ed[v]);
        }
    }
}

template <typename T>
static int cu_tet4_laplacian_diag_tpl(const ptrdiff_t nelements,
                                      const idx_t *const SFEM_RESTRICT elements,
                                      const cu_jacobian_t *const SFEM_RESTRICT fff,
                                      T *const diag,
                                      void *stream) {
    // Hand tuned
    int block_size = 128;
#ifdef SFEM_USE_OCCUPANCY_MAX_POTENTIAL
    {
        int min_grid_size;
        hipOccupancyMaxPotentialBlockSize(
                &min_grid_size, &block_size, cu_tet4_laplacian_diag_kernel<T>, 0, 0);
    }
#endif  // SFEM_USE_OCCUPANCY_MAX_POTENTIAL

    ptrdiff_t n_blocks = MAX(ptrdiff_t(1), (nelements + block_size - 1) / block_size);

    if (stream) {
        hipStream_t s = *static_cast<hipStream_t *>(stream);
        cu_tet4_laplacian_diag_kernel<<<n_blocks, block_size, 0, s>>>(
                nelements, elements, fff, diag);
    } else {
        cu_tet4_laplacian_diag_kernel<<<n_blocks, block_size, 0>>>(nelements, elements, fff, diag);
    }
    return 0;
}

extern int cu_tet4_laplacian_diag(const ptrdiff_t nelements,
                                  const idx_t *const SFEM_RESTRICT elements,
                                  const void *const SFEM_RESTRICT fff,
                                  const enum RealType real_type_xy_diag,
                                  void *const diag,
                                  void *stream) {
    switch (real_type_xy_diag) {
        case SFEM_REAL_DEFAULT: {
            return cu_tet4_laplacian_diag_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (real_t *)diag, stream);
        }
        case SFEM_FLOAT32: {
            return cu_tet4_laplacian_diag_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (float *)diag, stream);
        }
        case SFEM_FLOAT64: {
            return cu_tet4_laplacian_diag_tpl(
                    nelements, elements, (cu_jacobian_t *)fff, (double *)diag, stream);
        }
        default: {
            fprintf(stderr,
                    "[Error] cu_tet4_fff_fill: not implemented for type %s (code %d)\n",
                    real_type_to_string(real_type_xy_diag),
                    real_type_xy_diag);
            assert(0);
            return SFEM_FAILURE;
        }
    }
}
